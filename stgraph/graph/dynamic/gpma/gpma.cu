#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <hipcub/hipcub.hpp>

#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <iomanip>
#include <string>
#include <vector>
#include <tuple>
#include <chrono>
#include "stdio.h"

namespace py = pybind11;
using namespace pybind11::literals;

#define cErr(errcode)                             \
    {                                             \
        gpuAssert((errcode), __FILE__, __LINE__); \
    }
__inline__ __host__ __device__ void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

typedef unsigned long long KEY_TYPE;
typedef unsigned int VALUE_TYPE;
typedef unsigned int SIZE_TYPE;

typedef thrust::device_vector<KEY_TYPE> DEV_VEC_KEY;
typedef thrust::device_vector<VALUE_TYPE> DEV_VEC_VALUE;
typedef thrust::device_vector<SIZE_TYPE> DEV_VEC_SIZE;

typedef KEY_TYPE *KEY_PTR;
typedef VALUE_TYPE *VALUE_PTR;

enum UpdateActionKind { UAK_INSERT , UAK_DELETE };

#define RAW_PTR(x) thrust::raw_pointer_cast((x).data())

const KEY_TYPE KEY_NONE = 0xFFFFFFFFFFFFFFFF;
const KEY_TYPE KEY_MAX = 0xFFFFFFFFFFFFFFFE;
const SIZE_TYPE SIZE_NONE = 0xFFFFFFFF;
const VALUE_TYPE VALUE_NONE = 0;
const KEY_TYPE COL_IDX_NONE = 0xFFFFFFFF;

const SIZE_TYPE MAX_BLOCKS_NUM = 96 * 8;
#define CALC_BLOCKS_NUM(ITEMS_PER_BLOCK, CALC_SIZE) min(MAX_BLOCKS_NUM, (CALC_SIZE - 1) / ITEMS_PER_BLOCK + 1)

class GPMA
{
public:
    DEV_VEC_SIZE row_offset; // row offset vector
    DEV_VEC_KEY keys;   // column indices vector
    DEV_VEC_VALUE values;   // edge IDs vector
    DEV_VEC_SIZE node_ids;   // node IDs vector

    // node and edge metadata
    SIZE_TYPE row_num; // number of nodes
    SIZE_TYPE edge_count = 0;

    // node degree vectors
    DEV_VEC_SIZE in_degree;
    DEV_VEC_SIZE out_degree;
    DEV_VEC_SIZE cum_out_degree; // cummulative out degree

    // pointers to pinned node degree vectors
    SIZE_TYPE* in_degree_host;
    SIZE_TYPE* out_degree_host;

    // pointers to reverse CSR
    SIZE_TYPE* bwd_row_offset = NULL;
    KEY_TYPE* bwd_keys = NULL;
    VALUE_TYPE* bwd_values = NULL;

    // (add|delete)_updates stores the add/delete updates of every timestamp
    // (add|delete)_updates_count maintains the number of additions/deletions
    // in every timestamp
    std::vector<KEY_TYPE *> add_updates;
    std::vector<int> add_updates_count;
    std::vector<KEY_TYPE *> delete_updates;
    std::vector<int> delete_updates_count;

    // GPMA helper datastructures
    SIZE_TYPE segment_length;
    SIZE_TYPE tree_height;
    double density_lower_thres_leaf = 0.08;
    double density_lower_thres_root = 0.42;
    double density_upper_thres_root = 0.84;
    double density_upper_thres_leaf = 0.92;
    thrust::host_vector<SIZE_TYPE> lower_element;
    thrust::host_vector<SIZE_TYPE> upper_element;

    GPMA();

    inline int get_size()
    {
        return keys.size();
    }
};

GPMA::GPMA()
{
}

__forceinline__ __host__ __device__
    SIZE_TYPE
    fls(SIZE_TYPE x)
{
    SIZE_TYPE r = 32;
    if (!x)
        return 0;
    if (!(x & 0xffff0000u))
        x <<= 16, r -= 16;
    if (!(x & 0xff000000u))
        x <<= 8, r -= 8;
    if (!(x & 0xf0000000u))
        x <<= 4, r -= 4;
    if (!(x & 0xc0000000u))
        x <<= 2, r -= 2;
    if (!(x & 0x80000000u))
        x <<= 1, r -= 1;
    return r;
}

template <typename T>
__global__ void memcpy_kernel(T *dest, const T *src, SIZE_TYPE size)
{
    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    for (SIZE_TYPE i = global_thread_id; i < size; i += block_offset)
    {
        dest[i] = src[i];
    }
}

template <typename T>
__global__ void memset_kernel(T *data, T value, SIZE_TYPE size)
{
    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    for (SIZE_TYPE i = global_thread_id; i < size; i += block_offset)
    {
        data[i] = value;
    }
}

__host__ void recalculate_density(GPMA &gpma)
{
    gpma.lower_element.resize(gpma.tree_height + 1);
    gpma.upper_element.resize(gpma.tree_height + 1);
    cErr(hipDeviceSynchronize());

    SIZE_TYPE level_length = gpma.segment_length;
    for (SIZE_TYPE i = 0; i <= gpma.tree_height; i++)
    {
        double density_lower = gpma.density_lower_thres_root + (gpma.density_lower_thres_leaf - gpma.density_lower_thres_root) * (gpma.tree_height - i) / gpma.tree_height;
        double density_upper = gpma.density_upper_thres_root + (gpma.density_upper_thres_leaf - gpma.density_upper_thres_root) * (gpma.tree_height - i) / gpma.tree_height;

        gpma.lower_element[i] = (SIZE_TYPE)ceil(density_lower * level_length);
        gpma.upper_element[i] = (SIZE_TYPE)floor(density_upper * level_length);

        // special trim for wrong threshold introduced by float-integer conversion
        if (0 < i)
        {
            gpma.lower_element[i] = max(gpma.lower_element[i], 2 * gpma.lower_element[i - 1]);
            gpma.upper_element[i] = min(gpma.upper_element[i], 2 * gpma.upper_element[i - 1]);
        }
        level_length <<= 1;
    }
}

__device__ void cub_sort_key_value(KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE size, KEY_TYPE *tmp_keys,
                                   VALUE_TYPE *tmp_values)
{
    // Create a set of DoubleBuffers to wrap pairs of device pointers
    hipcub::DoubleBuffer<KEY_TYPE> d_keys(keys, tmp_keys);
    hipcub::DoubleBuffer<VALUE_TYPE> d_values(values, tmp_values);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    cErr(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, size));
    cErr(hipDeviceSynchronize());
    cErr(hipMalloc(&d_temp_storage, temp_storage_bytes));
    cErr(hipDeviceSynchronize());
    cErr(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, size));
    cErr(hipDeviceSynchronize());

    SIZE_TYPE THREADS_NUM = 128;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, size);
    memcpy_kernel<KEY_TYPE><<<BLOCKS_NUM, THREADS_NUM>>>(d_keys.Alternate(), d_keys.Current(), size);
    memcpy_kernel<VALUE_TYPE><<<BLOCKS_NUM, THREADS_NUM>>>(d_values.Alternate(), d_values.Current(), size);
    cErr(hipDeviceSynchronize());
    cErr(hipFree(d_temp_storage));
}

__device__ SIZE_TYPE handle_del_mod(KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE seg_length, KEY_TYPE key,
                                    VALUE_TYPE value, SIZE_TYPE leaf)
{

    if (VALUE_NONE == value)
        leaf = SIZE_NONE;
    for (SIZE_TYPE i = 0; i < seg_length; i++)
    {
        if (keys[i] == key)
        {
            values[i] = value;
            leaf = SIZE_NONE;
            break;
        }
    }
    return leaf;
}

__global__ void locate_leaf_kernel(KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE tree_size, SIZE_TYPE seg_length,
                                   SIZE_TYPE tree_height, KEY_TYPE *update_keys, VALUE_TYPE *update_values, SIZE_TYPE update_size,
                                   SIZE_TYPE *leaf)
{

    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    for (SIZE_TYPE i = global_thread_id; i < update_size; i += block_offset)
    {
        KEY_TYPE key = update_keys[i];
        VALUE_TYPE value = update_values[i];

        SIZE_TYPE prefix = 0;
        SIZE_TYPE current_bit = seg_length << tree_height >> 1;

        while (seg_length <= current_bit)
        {
            if (keys[prefix | current_bit] <= key)
                prefix |= current_bit;
            current_bit >>= 1;
        }

        prefix = handle_del_mod(keys + prefix, values + prefix, seg_length, key, value, prefix);
        leaf[i] = prefix;
    }
}

__host__ void locate_leaf_batch(KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE tree_size, SIZE_TYPE seg_length,
                                SIZE_TYPE tree_height, KEY_TYPE *update_keys, VALUE_TYPE *update_values, SIZE_TYPE update_size,
                                SIZE_TYPE *leaf)
{

    SIZE_TYPE THREADS_NUM = 32;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, update_size);

    locate_leaf_kernel<<<BLOCKS_NUM, THREADS_NUM>>>(keys, values, tree_size, seg_length, tree_height, update_keys,
                                                    update_values, update_size, leaf);
    cErr(hipDeviceSynchronize());
}

template <SIZE_TYPE THREAD_PER_BLOCK, SIZE_TYPE ITEM_PER_THREAD>
__device__ void block_compact_kernel(KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE &compacted_size)
{
    typedef hipcub::BlockScan<SIZE_TYPE, THREAD_PER_BLOCK> BlockScan;
    SIZE_TYPE thread_id = threadIdx.x;

    KEY_TYPE *block_keys = keys;
    VALUE_TYPE *block_values = values;

    KEY_TYPE thread_keys[ITEM_PER_THREAD];
    VALUE_TYPE thread_values[ITEM_PER_THREAD];

    SIZE_TYPE thread_offset = thread_id * ITEM_PER_THREAD;
    for (SIZE_TYPE i = 0; i < ITEM_PER_THREAD; i++)
    {
        thread_keys[i] = block_keys[thread_offset + i];
        thread_values[i] = block_values[thread_offset + i];
        block_keys[thread_offset + i] = KEY_NONE;
    }

    __shared__ typename BlockScan::TempStorage temp_storage;
    SIZE_TYPE thread_data[ITEM_PER_THREAD];
    for (SIZE_TYPE i = 0; i < ITEM_PER_THREAD; i++)
    {
        thread_data[i] = (thread_keys[i] == KEY_NONE || thread_values[i] == VALUE_NONE) ? 0 : 1;
    }
    __syncthreads();

    BlockScan(temp_storage).ExclusiveSum(thread_data, thread_data);
    __syncthreads();

    __shared__ SIZE_TYPE exscan[THREAD_PER_BLOCK * ITEM_PER_THREAD];
    for (SIZE_TYPE i = 0; i < ITEM_PER_THREAD; i++)
    {
        exscan[i + thread_offset] = thread_data[i];
    }
    __syncthreads();

    for (SIZE_TYPE i = 0; i < ITEM_PER_THREAD; i++)
    {
        if (thread_id == THREAD_PER_BLOCK - 1 && i == ITEM_PER_THREAD - 1)
            continue;
        if (exscan[thread_offset + i] != exscan[thread_offset + i + 1])
        {
            SIZE_TYPE loc = exscan[thread_offset + i];
            block_keys[loc] = thread_keys[i];
            block_values[loc] = thread_values[i];
        }
    }

    // special logic for the last element
    if (thread_id == THREAD_PER_BLOCK - 1)
    {
        SIZE_TYPE loc = exscan[THREAD_PER_BLOCK * ITEM_PER_THREAD - 1];
        if (thread_keys[ITEM_PER_THREAD - 1] == KEY_NONE || thread_values[ITEM_PER_THREAD - 1] == VALUE_NONE)
        {
            compacted_size = loc;
        }
        else
        {
            compacted_size = loc + 1;
            block_keys[loc] = thread_keys[ITEM_PER_THREAD - 1];
            block_values[loc] = thread_values[ITEM_PER_THREAD - 1];
        }
    }
}

template <typename FIRST_TYPE, typename SECOND_TYPE>
__device__ void block_pair_copy_kernel(FIRST_TYPE *dest_first, SECOND_TYPE *dest_second, FIRST_TYPE *src_first,
                                       SECOND_TYPE *src_second, SIZE_TYPE size)
{
    for (SIZE_TYPE i = threadIdx.x; i < size; i += blockDim.x)
    {
        dest_first[i] = src_first[i];
        dest_second[i] = src_second[i];
    }
}

template <SIZE_TYPE THREAD_PER_BLOCK, SIZE_TYPE ITEM_PER_THREAD>
__device__ void block_redispatch_kernel(KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE rebalance_width, SIZE_TYPE seg_length,
                                        SIZE_TYPE merge_size, SIZE_TYPE *row_offset, SIZE_TYPE update_node)
{

    // step1: load KV in shared memory
    __shared__ KEY_TYPE block_keys[THREAD_PER_BLOCK * ITEM_PER_THREAD];
    __shared__ VALUE_TYPE block_values[THREAD_PER_BLOCK * ITEM_PER_THREAD];
    block_pair_copy_kernel<KEY_TYPE, VALUE_TYPE>(block_keys, block_values, keys, values, rebalance_width);
    __syncthreads();

    // step2: sort by key with value on shared memory
    typedef hipcub::BlockLoad<KEY_TYPE, THREAD_PER_BLOCK, ITEM_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockKeyLoadT;
    typedef hipcub::BlockLoad<VALUE_TYPE, THREAD_PER_BLOCK, ITEM_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockValueLoadT;
    typedef hipcub::BlockStore<KEY_TYPE, THREAD_PER_BLOCK, ITEM_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockKeyStoreT;
    typedef hipcub::BlockStore<VALUE_TYPE, THREAD_PER_BLOCK, ITEM_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockValueStoreT;
    typedef hipcub::BlockRadixSort<KEY_TYPE, THREAD_PER_BLOCK, ITEM_PER_THREAD, VALUE_TYPE> BlockRadixSortT;

    __shared__ union
    {
        typename BlockKeyLoadT::TempStorage key_load;
        typename BlockValueLoadT::TempStorage value_load;
        typename BlockKeyStoreT::TempStorage key_store;
        typename BlockValueStoreT::TempStorage value_store;
        typename BlockRadixSortT::TempStorage sort;
    } temp_storage;

    KEY_TYPE thread_keys[ITEM_PER_THREAD];
    VALUE_TYPE thread_values[ITEM_PER_THREAD];
    BlockKeyLoadT(temp_storage.key_load).Load(block_keys, thread_keys);
    BlockValueLoadT(temp_storage.value_load).Load(block_values, thread_values);
    __syncthreads();

    BlockRadixSortT(temp_storage.sort).Sort(thread_keys, thread_values);
    __syncthreads();

    BlockKeyStoreT(temp_storage.key_store).Store(block_keys, thread_keys);
    BlockValueStoreT(temp_storage.value_store).Store(block_values, thread_values);
    __syncthreads();

    // step3: evenly re-dispatch KVs to leaf segments
    KEY_TYPE frac = rebalance_width / seg_length;
    KEY_TYPE deno = merge_size;
    for (SIZE_TYPE i = threadIdx.x; i < merge_size; i += blockDim.x)
    {
        keys[i] = KEY_NONE;
    }
    __syncthreads();

    for (SIZE_TYPE i = threadIdx.x; i < merge_size; i += blockDim.x)
    {
        SIZE_TYPE seg_idx = (SIZE_TYPE)(frac * i / deno);
        SIZE_TYPE seg_lane = (SIZE_TYPE)(frac * i % deno / frac);
        SIZE_TYPE proj_location = seg_idx * seg_length + seg_lane;

        KEY_TYPE cur_key = block_keys[i];
        VALUE_TYPE cur_value = block_values[i];
        keys[proj_location] = cur_key;
        values[proj_location] = cur_value;

        // addition for csr
        if ((cur_key & COL_IDX_NONE) == COL_IDX_NONE)
        {
            SIZE_TYPE cur_row = (SIZE_TYPE)(cur_key >> 32);
            row_offset[cur_row + 1] = proj_location + update_node;
        }
    }
}

template <SIZE_TYPE THREAD_PER_BLOCK, SIZE_TYPE ITEM_PER_THREAD>
__global__ void block_rebalancing_kernel(SIZE_TYPE seg_length, SIZE_TYPE level, KEY_TYPE *keys, VALUE_TYPE *values,
                                         SIZE_TYPE *update_nodes, KEY_TYPE *update_keys, VALUE_TYPE *update_values, SIZE_TYPE *unique_update_nodes,
                                         SIZE_TYPE *update_offset, SIZE_TYPE lower_bound, SIZE_TYPE upper_bound, SIZE_TYPE *row_offset)
{

    SIZE_TYPE update_id = blockIdx.x;
    SIZE_TYPE update_node = unique_update_nodes[update_id];
    KEY_TYPE *key = keys + update_node;
    VALUE_TYPE *value = values + update_node;
    SIZE_TYPE rebalance_width = seg_length << level;

    // compact
    __shared__ SIZE_TYPE compacted_size;
    block_compact_kernel<THREAD_PER_BLOCK, ITEM_PER_THREAD>(key, value, compacted_size);
    __syncthreads();

    // judge whether fit the density threshold
    SIZE_TYPE interval_a = update_offset[update_id];
    SIZE_TYPE interval_b = update_offset[update_id + 1];
    SIZE_TYPE interval_size = interval_b - interval_a;
    SIZE_TYPE merge_size = compacted_size + interval_size;
    __syncthreads();

    if (lower_bound <= merge_size && merge_size <= upper_bound)
    {
        // move
        block_pair_copy_kernel<KEY_TYPE, VALUE_TYPE>(key + compacted_size, value + compacted_size,
                                                     update_keys + interval_a, update_values + interval_a, interval_size);
        __syncthreads();

        // set SIZE_NONE for executed update
        for (SIZE_TYPE i = interval_a + threadIdx.x; i < interval_b; i += blockDim.x)
        {
            update_nodes[i] = SIZE_NONE;
        }

        // re-dispatch
        block_redispatch_kernel<THREAD_PER_BLOCK, ITEM_PER_THREAD>(key, value, rebalance_width, seg_length,
                                                                   merge_size, row_offset, update_node);
    }
}

__global__ void label_key_whether_none_kernel(SIZE_TYPE *label, KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE size)
{
    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    for (SIZE_TYPE i = global_thread_id; i < size; i += block_offset)
    {
        label[i] = (keys[i] == KEY_NONE || values[i] == VALUE_NONE) ? 0 : 1;
    }
}

__global__ void copy_compacted_kv(SIZE_TYPE *exscan, KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE size, KEY_TYPE *tmp_keys,
                                  VALUE_TYPE *tmp_values, SIZE_TYPE *compacted_size)
{

    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    for (SIZE_TYPE i = global_thread_id; i < size; i += block_offset)
    {
        if (i == size - 1)
            continue;
        if (exscan[i] != exscan[i + 1])
        {
            SIZE_TYPE loc = exscan[i];
            tmp_keys[loc] = keys[i];
            tmp_values[loc] = values[i];
        }
    }

    if (0 == global_thread_id)
    {
        SIZE_TYPE loc = exscan[size - 1];
        if (keys[size - 1] == KEY_NONE || values[size - 1] == VALUE_NONE)
        {
            *compacted_size = loc;
        }
        else
        {
            *compacted_size = loc + 1;
            tmp_keys[loc] = keys[size - 1];
            tmp_values[loc] = values[size - 1];
        }
    }
}

__device__ void compact_kernel(SIZE_TYPE size, KEY_TYPE *keys, VALUE_TYPE *values, SIZE_TYPE *compacted_size,
                               KEY_TYPE *tmp_keys, VALUE_TYPE *tmp_values, SIZE_TYPE *exscan, SIZE_TYPE *label)
{

    SIZE_TYPE THREADS_NUM = 32;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, size);
    label_key_whether_none_kernel<<<BLOCKS_NUM, THREADS_NUM>>>(label, keys, values, size);
    cErr(hipDeviceSynchronize());

    // exscan
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    cErr(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, label, exscan, size));
    cErr(hipDeviceSynchronize());
    cErr(hipMalloc(&d_temp_storage, temp_storage_bytes));
    cErr(hipDeviceSynchronize());
    cErr(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, label, exscan, size));
    cErr(hipDeviceSynchronize());
    cErr(hipFree(d_temp_storage));

    // copy compacted kv to tmp, and set the original to none
    copy_compacted_kv<<<BLOCKS_NUM, THREADS_NUM>>>(exscan, keys, values, size, tmp_keys, tmp_values, compacted_size);
    cErr(hipDeviceSynchronize());
}

__global__ void redispatch_kernel(KEY_TYPE *tmp_keys, VALUE_TYPE *tmp_values, KEY_TYPE *keys, VALUE_TYPE *values,
                                  SIZE_TYPE update_width, SIZE_TYPE seg_length, SIZE_TYPE merge_size, SIZE_TYPE *row_offset,
                                  SIZE_TYPE update_node)
{

    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    KEY_TYPE frac = update_width / seg_length;
    KEY_TYPE deno = merge_size;

    for (SIZE_TYPE i = global_thread_id; i < merge_size; i += block_offset)
    {
        SIZE_TYPE seg_idx = (SIZE_TYPE)(frac * i / deno);
        SIZE_TYPE seg_lane = (SIZE_TYPE)(frac * i % deno / frac);
        SIZE_TYPE proj_location = seg_idx * seg_length + seg_lane;
        KEY_TYPE cur_key = tmp_keys[i];
        VALUE_TYPE cur_value = tmp_values[i];
        keys[proj_location] = cur_key;
        values[proj_location] = cur_value;

        // addition for csr
        if ((cur_key & COL_IDX_NONE) == COL_IDX_NONE)
        {
            SIZE_TYPE cur_row = (SIZE_TYPE)(cur_key >> 32);
            row_offset[cur_row + 1] = proj_location + update_node;
        }
    }
}

__global__ void rebalancing_kernel(SIZE_TYPE unique_update_size, SIZE_TYPE seg_length, SIZE_TYPE level, KEY_TYPE *keys,
                                   VALUE_TYPE *values, SIZE_TYPE *update_nodes, KEY_TYPE *update_keys, VALUE_TYPE *update_values,
                                   SIZE_TYPE *unique_update_nodes, SIZE_TYPE *update_offset, SIZE_TYPE lower_bound, SIZE_TYPE upper_bound,
                                   SIZE_TYPE *row_offset, KEY_TYPE* tmp_keys, VALUE_TYPE* tmp_values, SIZE_TYPE* tmp_exscan, SIZE_TYPE* tmp_label)
{

    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;
    SIZE_TYPE update_width = seg_length << level;

    SIZE_TYPE *compacted_size;
    cErr(hipMalloc(&compacted_size, sizeof(SIZE_TYPE)));
    cErr(hipDeviceSynchronize());

    tmp_keys = tmp_keys + (blockIdx.x * update_width);
    tmp_values = tmp_values + (blockIdx.x * update_width);
    tmp_exscan = tmp_exscan + (blockIdx.x * update_width);
    tmp_label = tmp_label + (blockIdx.x * update_width);

    for (SIZE_TYPE i = global_thread_id; i < unique_update_size; i += block_offset)
    {
        SIZE_TYPE update_node = unique_update_nodes[i];
        KEY_TYPE *key = keys + update_node;
        VALUE_TYPE *value = values + update_node;

        // compact
        compact_kernel(update_width, key, value, compacted_size, tmp_keys, tmp_values, tmp_exscan, tmp_label);
        cErr(hipDeviceSynchronize());

        // judge whether fit the density threshold
        SIZE_TYPE interval_a = update_offset[i];
        SIZE_TYPE interval_b = update_offset[i + 1];
        SIZE_TYPE interval_size = interval_b - interval_a;
        SIZE_TYPE merge_size = (*compacted_size) + interval_size;

        if (lower_bound <= merge_size && merge_size <= upper_bound)
        {
            SIZE_TYPE THREADS_NUM = 32;
            SIZE_TYPE BLOCKS_NUM;

            // move
            BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, interval_size);
            memcpy_kernel<KEY_TYPE><<<BLOCKS_NUM, THREADS_NUM>>>(tmp_keys + (*compacted_size),
                                                                 update_keys + interval_a, interval_size);
            memcpy_kernel<VALUE_TYPE><<<BLOCKS_NUM, THREADS_NUM>>>(tmp_values + (*compacted_size),
                                                                   update_values + interval_a, interval_size);
            cErr(hipDeviceSynchronize());

            // set SIZE_NONE for executed updates
            memset_kernel<SIZE_TYPE><<<BLOCKS_NUM, THREADS_NUM>>>(update_nodes + interval_a, SIZE_NONE, interval_size);
            cErr(hipDeviceSynchronize());

            cub_sort_key_value(tmp_keys, tmp_values, merge_size, key, value);

            // re-dispatch
            BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, update_width);
            memset_kernel<KEY_TYPE><<<BLOCKS_NUM, THREADS_NUM>>>(key, KEY_NONE, update_width);
            cErr(hipDeviceSynchronize());

            BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, merge_size);
            redispatch_kernel<<<BLOCKS_NUM, THREADS_NUM>>>(tmp_keys, tmp_values, key, value, update_width, seg_length,
                                                           merge_size, row_offset, update_node);
            cErr(hipDeviceSynchronize());
        }

        cErr(hipDeviceSynchronize());
    }

    cErr(hipFree(compacted_size));
}

__host__ void rebalance_batch(SIZE_TYPE level, SIZE_TYPE seg_length, KEY_TYPE *keys, VALUE_TYPE *values,
                              SIZE_TYPE *update_nodes, KEY_TYPE *update_keys, VALUE_TYPE *update_values, SIZE_TYPE update_size,
                              SIZE_TYPE *unique_update_nodes, SIZE_TYPE *update_offset, SIZE_TYPE unique_update_size,
                              SIZE_TYPE lower_bound, SIZE_TYPE upper_bound, SIZE_TYPE *row_offset)
{

    SIZE_TYPE update_width = seg_length << level;

    if (update_width <= 1024)
    {
        // func pointer for each template
        void (*func_arr[10])(SIZE_TYPE, SIZE_TYPE, KEY_TYPE *, VALUE_TYPE *, SIZE_TYPE *, KEY_TYPE *, VALUE_TYPE *,
                             SIZE_TYPE *, SIZE_TYPE *, SIZE_TYPE, SIZE_TYPE, SIZE_TYPE *);
        func_arr[0] = block_rebalancing_kernel<2, 1>;
        func_arr[1] = block_rebalancing_kernel<4, 1>;
        func_arr[2] = block_rebalancing_kernel<8, 1>;
        func_arr[3] = block_rebalancing_kernel<16, 1>;
        func_arr[4] = block_rebalancing_kernel<32, 1>;
        func_arr[5] = block_rebalancing_kernel<32, 2>;
        func_arr[6] = block_rebalancing_kernel<32, 4>;
        func_arr[7] = block_rebalancing_kernel<32, 8>;
        func_arr[8] = block_rebalancing_kernel<32, 16>;
        func_arr[9] = block_rebalancing_kernel<32, 32>;

        // operate each tree node by cuda-block
        SIZE_TYPE THREADS_NUM = update_width > 32 ? 32 : update_width;
        SIZE_TYPE BLOCKS_NUM = unique_update_size;

        func_arr[fls(update_width) - 2]<<<BLOCKS_NUM, THREADS_NUM>>>(seg_length, level, keys, values, update_nodes,
                                                                     update_keys, update_values, unique_update_nodes, update_offset, lower_bound, upper_bound, row_offset);
        
        cErr(hipDeviceSynchronize());
    }
    else
    {
        // operate each tree node by cub-kernel (dynamic parallelsim)
        SIZE_TYPE BLOCKS_NUM = min(2048, unique_update_size);

        KEY_TYPE *tmp_keys;
        VALUE_TYPE *tmp_values;
        SIZE_TYPE *tmp_exscan;
        SIZE_TYPE *tmp_label;
        cErr(hipMalloc(&tmp_keys, BLOCKS_NUM * update_width * sizeof(KEY_TYPE)));
        cErr(hipMalloc(&tmp_values, BLOCKS_NUM * update_width * sizeof(VALUE_TYPE)));
        cErr(hipMalloc(&tmp_exscan, BLOCKS_NUM * update_width * sizeof(SIZE_TYPE)));
        cErr(hipMalloc(&tmp_label, BLOCKS_NUM * update_width * sizeof(SIZE_TYPE)));
        cErr(hipDeviceSynchronize());

        rebalancing_kernel<<<BLOCKS_NUM, 1>>>(unique_update_size, seg_length, level, keys, values, update_nodes,
                                              update_keys, update_values, unique_update_nodes, update_offset, lower_bound, upper_bound, row_offset, tmp_keys, tmp_values, tmp_exscan, tmp_label);
        
        cErr(hipDeviceSynchronize());
        cErr(hipFree(tmp_keys));
        cErr(hipFree(tmp_values));
        cErr(hipFree(tmp_exscan));
        cErr(hipFree(tmp_label));
    }
    
}

struct three_tuple_first_none
{
    typedef thrust::tuple<SIZE_TYPE, KEY_TYPE, VALUE_TYPE> Tuple;
    __host__ __device__ bool operator()(const Tuple &a)
    {
        return SIZE_NONE == thrust::get<0>(a);
    }
};
__host__ void compact_insertions(DEV_VEC_SIZE &update_nodes, DEV_VEC_KEY &update_keys, DEV_VEC_VALUE &update_values,
                                 SIZE_TYPE &update_size)
{

    auto zip_begin = thrust::make_zip_iterator(
        thrust::make_tuple(update_nodes.begin(), update_keys.begin(), update_values.begin()));
    auto zip_end = thrust::remove_if(zip_begin, zip_begin + update_size, three_tuple_first_none());
    cErr(hipDeviceSynchronize());
    update_size = zip_end - zip_begin;
}

__host__ SIZE_TYPE group_insertion_by_node(SIZE_TYPE *update_nodes, SIZE_TYPE update_size,
                                           SIZE_TYPE *unique_update_nodes, SIZE_TYPE *update_offset)
{

    // step1: encode
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    SIZE_TYPE *tmp_offset;
    cErr(hipMalloc(&tmp_offset, sizeof(SIZE_TYPE) * update_size));

    SIZE_TYPE *num_runs_out;
    cErr(hipMalloc(&num_runs_out, sizeof(SIZE_TYPE)));
    cErr(hipDeviceSynchronize());
    cErr(hipcub::DeviceRunLengthEncode::Encode(d_temp_storage, temp_storage_bytes, update_nodes,
                                            unique_update_nodes, tmp_offset, num_runs_out, update_size));
    cErr(hipDeviceSynchronize());
    cErr(hipMalloc(&d_temp_storage, temp_storage_bytes));
    cErr(hipDeviceSynchronize());
    cErr(hipcub::DeviceRunLengthEncode::Encode(d_temp_storage, temp_storage_bytes, update_nodes,
                                            unique_update_nodes, tmp_offset, num_runs_out, update_size));
    cErr(hipDeviceSynchronize());

    SIZE_TYPE unique_node_size[1];
    cErr(hipMemcpy(unique_node_size, num_runs_out, sizeof(SIZE_TYPE), hipMemcpyDeviceToHost));
    cErr(hipDeviceSynchronize());
    cErr(hipFree(num_runs_out));
    cErr(hipFree(d_temp_storage));

    // step2: exclusive scan
    d_temp_storage = NULL;
    temp_storage_bytes = 0;
    cErr(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, tmp_offset,
                                       update_offset, unique_node_size[0]));
    cErr(hipDeviceSynchronize());
    cErr(hipMalloc(&d_temp_storage, temp_storage_bytes));
    cErr(hipDeviceSynchronize());
    cErr(hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, tmp_offset,
                                       update_offset, unique_node_size[0]));
    cErr(hipDeviceSynchronize());
    cErr(hipFree(d_temp_storage));

    cErr(hipMemcpy(update_offset + unique_node_size[0], &update_size, sizeof(SIZE_TYPE), hipMemcpyHostToDevice));
    cErr(hipDeviceSynchronize());
    cErr(hipFree(tmp_offset));

    return unique_node_size[0];
}

__host__ void compress_insertions_by_node(DEV_VEC_SIZE &update_nodes, SIZE_TYPE update_size,
                                          DEV_VEC_SIZE &unique_update_nodes, DEV_VEC_SIZE &update_offset, SIZE_TYPE &unique_node_size)
{
    unique_node_size = group_insertion_by_node(RAW_PTR(update_nodes), update_size, RAW_PTR(unique_update_nodes),
                                               RAW_PTR(update_offset));
    cErr(hipDeviceSynchronize());
}

__global__ void up_level_kernel(SIZE_TYPE *update_nodes, SIZE_TYPE update_size, SIZE_TYPE update_width)
{
    SIZE_TYPE global_thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    SIZE_TYPE block_offset = gridDim.x * blockDim.x;

    for (SIZE_TYPE i = global_thread_id; i < update_size; i += block_offset)
    {
        SIZE_TYPE node = update_nodes[i];
        update_nodes[i] = node & ~update_width;
    }
}

__host__ void up_level_batch(SIZE_TYPE *update_nodes, SIZE_TYPE update_size, SIZE_TYPE update_width)
{
    SIZE_TYPE THREADS_NUM = 32;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, update_size);
    up_level_kernel<<<BLOCKS_NUM, THREADS_NUM>>>(update_nodes, update_size, update_width);
    cErr(hipDeviceSynchronize());
}

struct kv_tuple_none
{
    typedef thrust::tuple<KEY_TYPE, VALUE_TYPE> Tuple;
    __host__ __device__ bool operator()(const Tuple &a)
    {
        return KEY_NONE == thrust::get<0>(a) || VALUE_NONE == thrust::get<1>(a);
    }
};
__host__ int resize_gpma(GPMA &gpma, DEV_VEC_KEY &update_keys, DEV_VEC_VALUE &update_values, SIZE_TYPE update_size)
{
    auto zip_begin = thrust::make_zip_iterator(thrust::make_tuple(gpma.keys.begin(), gpma.values.begin()));
    auto zip_end = thrust::remove_if(zip_begin, zip_begin + gpma.keys.size(), kv_tuple_none());
    cErr(hipDeviceSynchronize());
    SIZE_TYPE compacted_size = zip_end - zip_begin;
    thrust::fill(gpma.keys.begin() + compacted_size, gpma.keys.end(), KEY_NONE);
    cErr(hipDeviceSynchronize());

    SIZE_TYPE merge_size = compacted_size + update_size;
    SIZE_TYPE original_tree_size = gpma.keys.size();

    SIZE_TYPE tree_size = 4;
    while (floor(gpma.density_upper_thres_root * tree_size) < merge_size)
        tree_size <<= 1;
    gpma.segment_length = 1 << (fls(fls(tree_size)) - 1);
    gpma.tree_height = fls(tree_size / gpma.segment_length) - 1;

    gpma.keys.resize(tree_size, KEY_NONE);
    gpma.values.resize(tree_size);
    cErr(hipDeviceSynchronize());
    recalculate_density(gpma);

    return compacted_size;
}

__host__ void significant_insert(GPMA &gpma, DEV_VEC_KEY &update_keys, DEV_VEC_VALUE &update_values, int update_size)
{
    int valid_size = resize_gpma(gpma, update_keys, update_values, update_size);
    thrust::copy(update_keys.begin(), update_keys.begin() + update_size, gpma.keys.begin() + valid_size);
    thrust::copy(update_values.begin(), update_values.begin() + update_size, gpma.values.begin() + valid_size);

    DEV_VEC_KEY tmp_update_keys(gpma.get_size());
    DEV_VEC_VALUE tmp_update_values(gpma.get_size());
    cErr(hipDeviceSynchronize());

    int merge_size = valid_size + update_size;
    thrust::sort_by_key(gpma.keys.begin(), gpma.keys.begin() + merge_size, gpma.values.begin());
    cErr(hipDeviceSynchronize());

    SIZE_TYPE THREADS_NUM = 32;
    SIZE_TYPE BLOCKS_NUM;
    BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, merge_size);
    redispatch_kernel<<<BLOCKS_NUM, THREADS_NUM>>>(RAW_PTR(gpma.keys), RAW_PTR(gpma.values), RAW_PTR(tmp_update_keys),
                                                   RAW_PTR(tmp_update_values), gpma.get_size(), gpma.segment_length, merge_size, RAW_PTR(gpma.row_offset), 0);
    cErr(hipDeviceSynchronize());

    gpma.keys = tmp_update_keys;
    gpma.values = tmp_update_values;
    cErr(hipDeviceSynchronize());
}

__host__ void update_gpma(GPMA &gpma, DEV_VEC_KEY &update_keys, DEV_VEC_VALUE &update_values)
{
    SIZE_TYPE ous = update_keys.size();

    // step1: sort update keys with values
    // COMMENTING THIS OUT SINCE WE ARE PASSING IN SORTED INPUT, UNCOMMENT OTHERWISE
    // thrust::sort_by_key(update_keys.begin(), update_keys.end(), update_values.begin());
    // cErr(hipDeviceSynchronize());

    // step2: get leaf node of each update (execute del and mod)
    DEV_VEC_SIZE update_nodes(update_keys.size());
    cErr(hipDeviceSynchronize());
    locate_leaf_batch(RAW_PTR(gpma.keys), RAW_PTR(gpma.values), gpma.keys.size(), gpma.segment_length, gpma.tree_height,
                      RAW_PTR(update_keys), RAW_PTR(update_values), update_keys.size(), RAW_PTR(update_nodes));
    cErr(hipDeviceSynchronize());

    // step3: extract insertions
    DEV_VEC_SIZE unique_update_nodes(update_keys.size());
    DEV_VEC_SIZE update_offset(update_keys.size() + 1);
    cErr(hipDeviceSynchronize());
    SIZE_TYPE update_size = update_nodes.size();
    SIZE_TYPE unique_node_size = 0;
    compact_insertions(update_nodes, update_keys, update_values, update_size);
    compress_insertions_by_node(update_nodes, update_size, unique_update_nodes, update_offset, unique_node_size);
    cErr(hipDeviceSynchronize());

    // step4: rebuild for significant update
    int threshold = 5 * 1000 * 1000;
    if (update_size >= threshold)
    {
        significant_insert(gpma, update_keys, update_values, update_size);
        return;
    }

    // step5: rebalance each tree level
    for (SIZE_TYPE level = 0; level <= gpma.tree_height && update_size; level++)
    {
        SIZE_TYPE lower_bound = gpma.lower_element[level];
        SIZE_TYPE upper_bound = gpma.upper_element[level];

        // re-balance
        rebalance_batch(level, gpma.segment_length, RAW_PTR(gpma.keys), RAW_PTR(gpma.values), RAW_PTR(update_nodes),
                        RAW_PTR(update_keys), RAW_PTR(update_values), update_size, RAW_PTR(unique_update_nodes),
                        RAW_PTR(update_offset), unique_node_size, lower_bound, upper_bound, RAW_PTR(gpma.row_offset));

        // compact
        compact_insertions(update_nodes, update_keys, update_values, update_size);

        // up level
        up_level_batch(RAW_PTR(update_nodes), update_size, gpma.segment_length << level);

        // re-compress
        compress_insertions_by_node(update_nodes, update_size, unique_update_nodes, update_offset,
                                    unique_node_size);
    }

    // step6: rebalance the root node if necessary
    if (update_size > 0)
    {
        resize_gpma(gpma, update_keys, update_values, update_size);

        SIZE_TYPE level = gpma.tree_height;
        SIZE_TYPE lower_bound = gpma.lower_element[level];
        SIZE_TYPE upper_bound = gpma.upper_element[level];

        // re-balance
        cErr(hipDeviceSynchronize());
        rebalance_batch(level, gpma.segment_length, RAW_PTR(gpma.keys), RAW_PTR(gpma.values), RAW_PTR(update_nodes),
                        RAW_PTR(update_keys), RAW_PTR(update_values), update_size, RAW_PTR(unique_update_nodes),
                        RAW_PTR(update_offset), unique_node_size, lower_bound, upper_bound, RAW_PTR(gpma.row_offset));
    }

    cErr(hipDeviceSynchronize());
}

__host__ void build_gpma(GPMA &gpma)
{
    // we are creating the smallest possible GPMA tree
    // height = 1
    // levels = 2
    // leaf segment length = 2
    gpma.keys.resize(4, KEY_NONE);
    gpma.values.resize(4);

    cErr(hipDeviceSynchronize());

    gpma.segment_length = 2;
    gpma.tree_height = 1;

    // the minimal tree structure has 2 levels with 4 elements' space, and the leaf segment's length is 2
    // put two MAX_KEY to keep minimal valid structure
    gpma.keys[0] = gpma.keys[2] = KEY_MAX;
    gpma.values[0] = gpma.values[2] = 1;

    recalculate_density(gpma);
}

template <typename T>
struct col_idx_none
{
    typedef T argument_type;
    typedef T result_type;
    __host__ __device__
        T
        operator()(const T &x) const
    {
        return (x << 32) + COL_IDX_NONE;
    }
};
__host__ void init_gpma(GPMA &gpma, SIZE_TYPE row_num)
{
    // gpma     (GPMA)          : The GPMA object whose CSR arrays are to initialised
    // row_num  (unsized int)   : Number of total nodes in the graph

    // initialising the row_offset vector with all 0 value
    gpma.row_num = row_num;
    gpma.row_offset.resize(row_num + 1, 0);
    gpma.node_ids.resize(row_num, 0);

    // initialising in_degree, out_degree and cum_out_degree arrays
    // with all zero values
    gpma.in_degree.resize(row_num, 0);
    gpma.out_degree.resize(row_num, 0);
    gpma.cum_out_degree.resize(row_num, 0);

    // allocating pinned memory for node degree vectors
    cErr(hipHostMalloc(&gpma.in_degree_host, sizeof(SIZE_TYPE) * row_num));
    cErr(hipHostMalloc(&gpma.out_degree_host, sizeof(SIZE_TYPE) * row_num));

    // creates a device vector of size row_num
    DEV_VEC_KEY row_wall(row_num);

    // creates a device vector of size row_num all
    // initialised to the value 1
    DEV_VEC_VALUE tmp_value(row_num, 1);
    cErr(hipDeviceSynchronize());

    thrust::tabulate(row_wall.begin(), row_wall.end(), col_idx_none<KEY_TYPE>());
    build_gpma(gpma);
    cErr(hipDeviceSynchronize());
    update_gpma(gpma, row_wall, tmp_value);
    cErr(hipDeviceSynchronize());
}

//////////////////////////////////////// GPMA APIs ///////////////////////////////////////////////////////////

void init_graph_updates(GPMA &gpma, std::map<std::string, std::map<std::string, std::vector<std::tuple<int, int>>>> updates, bool reverse_edges = false)
{

    gpma.add_updates.resize(updates.size());
    gpma.delete_updates.resize(updates.size());
    gpma.add_updates_count.resize(updates.size());
    gpma.delete_updates_count.resize(updates.size());

    std::vector<std::tuple<int, int>> update_tup;

    for (int t = 0; t < updates.size(); ++t)
    {
        // preparing keys of edges to be added
        update_tup = updates[std::to_string(t)]["add"];
        std::vector<KEY_TYPE> add_key(update_tup.size());
        for (int i = 0; i < update_tup.size(); ++i)
        {
            unsigned int src = reverse_edges ? std::get<1>(update_tup[i]) : std::get<0>(update_tup[i]);
            unsigned int dst = reverse_edges ? std::get<0>(update_tup[i]) : std::get<1>(update_tup[i]);
            add_key[i] = ((KEY_TYPE)src << 32) + dst;
        }

        // preparing keys of edges to be deleted
        update_tup = updates[std::to_string(t)]["delete"];
        std::vector<KEY_TYPE> delete_key(update_tup.size());
        for (int i = 0; i < update_tup.size(); ++i)
        {
            unsigned int src = reverse_edges ? std::get<1>(update_tup[i]) : std::get<0>(update_tup[i]);
            unsigned int dst = reverse_edges ? std::get<0>(update_tup[i]) : std::get<1>(update_tup[i]);
            delete_key[i] = ((KEY_TYPE)src << 32) + dst;
        }

        // allocating and copying to GPU memory
        KEY_TYPE *gpu_add_key;
        KEY_TYPE *gpu_delete_key;
        cErr(hipMalloc(&gpu_add_key, sizeof(KEY_TYPE) * add_key.size()));
        cErr(hipMalloc(&gpu_delete_key, sizeof(KEY_TYPE) * delete_key.size()));
        cErr(hipMemcpy(gpu_add_key, add_key.data(), sizeof(KEY_TYPE) * add_key.size(), hipMemcpyHostToDevice));
        cErr(hipMemcpy(gpu_delete_key, delete_key.data(), sizeof(KEY_TYPE) * delete_key.size(), hipMemcpyHostToDevice));

        // storing the pointers and counts in GPMA object
        gpma.add_updates[t] = gpu_add_key;
        gpma.delete_updates[t] = gpu_delete_key;
        gpma.add_updates_count[t] = updates[std::to_string(t)]["add"].size();
        gpma.delete_updates_count[t] = updates[std::to_string(t)]["delete"].size();
    }
}

template <UpdateActionKind Action>
__global__ void update_node_degrees_kernel(SIZE_TYPE* in_degree, SIZE_TYPE* out_degree, KEY_TYPE* updates, int updates_size){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int block_offset = gridDim.x * blockDim.x;
    for(; index < updates_size; index += block_offset){
        KEY_TYPE key = updates[index];
        SIZE_TYPE src = (SIZE_TYPE)(key >> 32);
        SIZE_TYPE dst = (SIZE_TYPE)(key);

        if(Action == UAK_INSERT){
            atomicAdd(&in_degree[dst],1);
            atomicAdd(&out_degree[src],1);
        } else if (Action == UAK_DELETE) {
            atomicSub(&in_degree[dst],1);
            atomicSub(&out_degree[src],1);
        }
    }
}

void update_node_degrees(SIZE_TYPE* in_degree, SIZE_TYPE* out_degree, KEY_TYPE* add_keys, int add_key_size, KEY_TYPE* del_keys, int del_key_size){
    
    // Updating node degrees associated with added edges
    SIZE_TYPE THREADS_NUM = 128;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, (SIZE_TYPE)add_key_size);
    update_node_degrees_kernel<UAK_INSERT> <<<BLOCKS_NUM,THREADS_NUM>>>(in_degree, out_degree, add_keys, add_key_size);

    // Updating node degrees associated with deleted edges
    BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, (SIZE_TYPE)del_key_size);
    update_node_degrees_kernel<UAK_DELETE> <<<BLOCKS_NUM,THREADS_NUM>>>(in_degree, out_degree, del_keys, del_key_size);
    cErr(hipDeviceSynchronize());
}

std::vector<float> edge_update_t(GPMA &gpma, int timestamp, bool revert_update = false)
{   
    // This function performs/reverts the updates of a given timestamp
    // based on if revert_update is false or true respectively.

    int add_edge_count, delete_edge_count;
    KEY_TYPE *add_updates_ptr, *delete_updates_ptr;

    if (revert_update)
    {
        add_edge_count = gpma.delete_updates_count[timestamp];
        delete_edge_count = gpma.add_updates_count[timestamp];
        add_updates_ptr = gpma.delete_updates[timestamp];
        delete_updates_ptr = gpma.add_updates[timestamp];
    }
    else
    {
        add_edge_count = gpma.add_updates_count[timestamp];
        delete_edge_count = gpma.delete_updates_count[timestamp];
        add_updates_ptr = gpma.add_updates[timestamp];
        delete_updates_ptr = gpma.delete_updates[timestamp];
    }
    gpma.edge_count = gpma.edge_count + add_edge_count - delete_edge_count;

    // Preparing data for addition updates
    thrust::device_ptr<KEY_TYPE> add_key_thrust_dev_ptr = thrust::device_pointer_cast(add_updates_ptr);
    DEV_VEC_KEY add_key_thrust_dev(add_key_thrust_dev_ptr, add_key_thrust_dev_ptr + add_edge_count);
    thrust::device_vector<VALUE_TYPE> add_value_thrust_dev(add_edge_count, 1);
    hipDeviceSynchronize();

    // Preparing data for deletion updates
    thrust::device_ptr<KEY_TYPE> delete_key_thrust_dev_ptr = thrust::device_pointer_cast(delete_updates_ptr);
    DEV_VEC_KEY delete_key_thrust_dev(delete_key_thrust_dev_ptr, delete_key_thrust_dev_ptr + delete_edge_count);
    thrust::device_vector<VALUE_TYPE> delete_value_thrust_dev(delete_edge_count, VALUE_NONE);
    hipDeviceSynchronize();

    // Performing GPMA update
    auto start_time_update = std::chrono::high_resolution_clock::now();
    update_gpma(gpma, add_key_thrust_dev, add_value_thrust_dev);
    hipDeviceSynchronize();
    update_gpma(gpma, delete_key_thrust_dev, delete_value_thrust_dev);
    hipDeviceSynchronize();
    auto end_time_update = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> time_update = (end_time_update - start_time_update);

    // Performing node degree update
    auto start_time_node_degrees = std::chrono::high_resolution_clock::now();
    update_node_degrees(RAW_PTR(gpma.in_degree), RAW_PTR(gpma.out_degree), add_updates_ptr, add_edge_count, delete_updates_ptr, delete_edge_count);
    auto end_time_node_degrees = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> time_node_degree = (end_time_node_degrees - start_time_node_degrees);

    std::vector<float> vec;
    vec.push_back(time_update.count());
    vec.push_back(time_node_degree.count());
    return vec;
}

__global__ void label_edges_kernel(SIZE_TYPE* row_offset, KEY_TYPE* column_indices, VALUE_TYPE* eids, SIZE_TYPE* cum_out_degree, int cum_out_degree_size){
    
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int block_offset = gridDim.x * blockDim.x;
    int edge_count = 1;
    int beg, end;

    for(; index < cum_out_degree_size; index += block_offset){
        beg = row_offset[index];
        end = row_offset[index+1];

        if(index > 0){
            edge_count = cum_out_degree[index-1]+1;
        }
        
        for(int i=beg; i<end; ++i){
            KEY_TYPE key = column_indices[i];
            VALUE_TYPE value = eids[i];
            if (key != KEY_MAX && (key & 0xffffffff) != COL_IDX_NONE && value != VALUE_NONE)
            {
                eids[i] = edge_count;
                ++edge_count;
            }
        }
    }
}

void label_edges(GPMA &gpma){
    // Calculating cummulative out degrees
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, RAW_PTR(gpma.out_degree), RAW_PTR(gpma.cum_out_degree), gpma.row_num);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, RAW_PTR(gpma.out_degree), RAW_PTR(gpma.cum_out_degree), gpma.row_num);
    cErr(hipDeviceSynchronize());
    cErr(hipFree(d_temp_storage));

    // Invoking the label edges kernel
    SIZE_TYPE THREADS_NUM = 128;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, (SIZE_TYPE)gpma.row_num);
    label_edges_kernel<<<BLOCKS_NUM,THREADS_NUM>>>(RAW_PTR(gpma.row_offset), RAW_PTR(gpma.keys), RAW_PTR(gpma.values), RAW_PTR(gpma.cum_out_degree), gpma.row_num);
    cErr(hipDeviceSynchronize());
}

__global__ void count_sort_kernel(SIZE_TYPE* bwd_row_offset, KEY_TYPE* bwd_column_indices, VALUE_TYPE* bwd_values, SIZE_TYPE* fwd_row_offset, KEY_TYPE* fwd_column_indices, VALUE_TYPE* fwd_values, int row_offset_size){
    // [WARNING 1] HERE (src,pos) WE ARE INDEXING WITH UNISGNED INT
    // THERE ARE POTENTIAL ISSUES THAT NEED TO BE CONSIDERED
    // [WARNING 2] THIS IS NO LONGER A STABLE SORT
    
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int block_offset = gridDim.x * blockDim.x;

    for(; index < row_offset_size-1; index += block_offset){
        int beg = fwd_row_offset[index];
        int end = fwd_row_offset[index+1];
        for(int i=beg; i<end; ++i){
            KEY_TYPE key = fwd_column_indices[i];
            VALUE_TYPE value = fwd_values[i];
            SIZE_TYPE src = key;
            if (key != KEY_MAX && src != COL_IDX_NONE && value != VALUE_NONE)
            {
                SIZE_TYPE pos = atomicSub(&bwd_row_offset[src],1) - 1;
                bwd_column_indices[pos] = ((KEY_TYPE)src << 32) + (key >> 32);
                bwd_values[pos] = value;
            }
        }
    }
}

std::vector<float> build_backward_csr(GPMA &gpma){

    // Step 1: Allocate space for backward CSR vectors
    auto start_time_alloc = std::chrono::high_resolution_clock::now();
    cErr(hipMalloc(&gpma.bwd_row_offset, (gpma.row_num+1) * sizeof(SIZE_TYPE)));
    cErr(hipMalloc(&gpma.bwd_keys, gpma.edge_count * sizeof(KEY_TYPE)));
    cErr(hipMalloc(&gpma.bwd_values, gpma.edge_count * sizeof(VALUE_TYPE)));
    auto end_time_alloc = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> time_alloc = (end_time_alloc - start_time_alloc);

    // Step 2: Find the cummulative sum of the in-degree vector
    auto start_time_cum = std::chrono::high_resolution_clock::now();
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, RAW_PTR(gpma.in_degree), gpma.bwd_row_offset, gpma.row_num);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, RAW_PTR(gpma.in_degree), gpma.bwd_row_offset, gpma.row_num);
    cErr(hipDeviceSynchronize());
    cErr(hipFree(d_temp_storage));

    // Step 3: Insert the edge count in the last element slot in bwd_row_offset
    SIZE_TYPE edge_count = gpma.edge_count;
    hipMemcpy(gpma.bwd_row_offset + gpma.row_num, &edge_count, sizeof(SIZE_TYPE), hipMemcpyHostToDevice);
    auto end_time_cum = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> time_cum = (end_time_cum - start_time_cum);

    // Step 4: Invoke the count_sort_kernel to perform the sort using the
    // last 32-bits as key
    auto start_time_count = std::chrono::high_resolution_clock::now();
    SIZE_TYPE THREADS_NUM = 128;
    SIZE_TYPE BLOCKS_NUM = CALC_BLOCKS_NUM(THREADS_NUM, (SIZE_TYPE)gpma.keys.size());
    count_sort_kernel<<<BLOCKS_NUM,THREADS_NUM>>>(gpma.bwd_row_offset, gpma.bwd_keys, gpma.bwd_values, RAW_PTR(gpma.row_offset), RAW_PTR(gpma.keys), RAW_PTR(gpma.values), gpma.row_offset.size());
    cErr(hipDeviceSynchronize());
    auto end_time_count = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> time_count = (end_time_count - start_time_count);

    std::vector<float> res;
    res.push_back(time_alloc.count());
    res.push_back(time_cum.count());
    res.push_back(time_count.count());
    return res;
}

void free_backward_csr(GPMA &gpma){
    cErr(hipFree(gpma.bwd_row_offset));
    cErr(hipFree(gpma.bwd_keys));
    cErr(hipFree(gpma.bwd_values));
}

std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t, std::uintptr_t> get_csr_ptrs(GPMA &gpma, bool is_backward = false)
{
    // This function returns CSR pointers to forward or backward graph
    // based on if is_backward is false or true respectively.

    std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t, std::uintptr_t> t;
    thrust::sequence(gpma.node_ids.begin(), gpma.node_ids.end());
    DEV_VEC_SIZE tmp(gpma.row_num);

    if(is_backward){
        thrust::copy(gpma.in_degree.begin(), gpma.in_degree.end(), tmp.begin());
        cErr(hipDeviceSynchronize());
        thrust::sort_by_key(tmp.begin(), tmp.end(), gpma.node_ids.begin(), thrust::greater<int>());
        cErr(hipDeviceSynchronize());

        std::get<0>(t) = (std::uintptr_t)gpma.bwd_row_offset;
        std::get<1>(t) = (std::uintptr_t)gpma.bwd_keys;
        std::get<2>(t) = (std::uintptr_t)gpma.bwd_values;
        std::get<3>(t) = (std::uintptr_t)RAW_PTR(gpma.node_ids);
    }else{
        thrust::copy(gpma.out_degree.begin(), gpma.out_degree.end(), tmp.begin());
        cErr(hipDeviceSynchronize());
        thrust::sort_by_key(tmp.begin(), tmp.end(), gpma.node_ids.begin(), thrust::greater<int>());
        cErr(hipDeviceSynchronize());

        std::get<0>(t) = (std::uintptr_t)RAW_PTR(gpma.row_offset);
        std::get<1>(t) = (std::uintptr_t)RAW_PTR(gpma.keys);
        std::get<2>(t) = (std::uintptr_t)RAW_PTR(gpma.values);
        std::get<3>(t) = (std::uintptr_t)RAW_PTR(gpma.node_ids);
    }
    return t;
}

std::vector<SIZE_TYPE> get_in_degrees(GPMA &gpma){
    cErr(hipMemcpy(gpma.in_degree_host, RAW_PTR(gpma.in_degree), sizeof(SIZE_TYPE) * gpma.row_num, hipMemcpyDeviceToHost));
    std::vector<SIZE_TYPE> in_degree(gpma.in_degree_host, gpma.in_degree_host + gpma.row_num);
    return in_degree;
} 

std::vector<SIZE_TYPE> get_out_degrees(GPMA &gpma){
    cErr(hipMemcpy(gpma.out_degree_host, RAW_PTR(gpma.out_degree), sizeof(SIZE_TYPE) * gpma.row_num, hipMemcpyDeviceToHost));
    std::vector<SIZE_TYPE> out_degree(gpma.out_degree_host, gpma.out_degree_host + gpma.row_num);
    return out_degree;
} 

//////////////////////////////////////// GPMA Logging APIs ///////////////////////////////////////////////////////////

std::tuple<int, int> get_graph_attr(GPMA &gpma)
{
    std::tuple<int, int> t;

    // Since row_offset has size num_nodes + 1
    std::get<0>(t) = gpma.row_offset.size() - 1;
    std::get<1>(t) = gpma.edge_count;
    return t;
}

void print_gpma_info(GPMA &gpma, int node)
{
    // Pretty prints the row_offset range and column_indices values
    // for the given node
    //
    // row_offset:  Row offset vector
    // col_indices: Column indices PMA
    // node:        Node whose row_offset and column_indices is to be printed

    // getting the (beg,end) range of the node

    thrust::host_vector<SIZE_TYPE> row_offset = gpma.row_offset;
    thrust::host_vector<KEY_TYPE> col_indices = gpma.keys;
    thrust::host_vector<VALUE_TYPE> edge_values = gpma.values;

    unsigned int beg = row_offset[node];
    unsigned int end = row_offset[node + 1];
    int print_count = 0;

    py::print("\n🌟 Column Indices for node: ", node, "(", beg, ", ", end, ")", "\n\n");
    // std::cout << std::setw(6);

    for (int i = beg; i < end; ++i)
    {
        // KEY_TYPE mask = (KEY_TYPE)node << 32;
        // unsigned int dst = (col_indices[i] - mask);
        unsigned int dst = (col_indices[i] & 0xffffffff);
        VALUE_TYPE val = edge_values[i];

        if (col_indices[i] != KEY_MAX && (col_indices[i] & COL_IDX_NONE != COL_IDX_NONE) && val != VALUE_NONE)
        {
            py::print("[", col_indices[i], "] ", dst, "(", val, ")", "  ");
        }
        else
        {
            py::print("[", col_indices[i], "] ", "(-)(", val, ")", "  ");
        }
        print_count += 1;

        if (print_count > 10)
        {
            py::print("\n");
            print_count = 0;
        }
    }
    py::print("\n\n");
}

std::vector<unsigned int> get_node_ids(GPMA &gpma){
    thrust::host_vector<SIZE_TYPE> h_node_ids = gpma.node_ids;
    std::vector<unsigned int> node_ids(h_node_ids.size());
    for(int i=0; i<h_node_ids.size(); ++i){
        node_ids[i] = h_node_ids[i];
    }
    return node_ids;
}

std::set<std::tuple<unsigned int, unsigned int, unsigned int>> get_gpma_edge_list(GPMA &gpma)
{

    std::set<std::tuple<unsigned int, unsigned int, unsigned int>> vec;
    thrust::host_vector<SIZE_TYPE> h_ref_row_offset = gpma.row_offset;
    thrust::host_vector<KEY_TYPE>
        h_ref_keys = gpma.keys;
    thrust::host_vector<VALUE_TYPE> h_ref_values = gpma.values;

    cErr(hipDeviceSynchronize());

    // py::print("ROW_OFFSET: ", h_ref_row_offset.size());
    // py::print("COL_IDX_SIZE: ", h_ref_keys.size());
    // py::print("VALUES_SIZE: ", h_ref_values.size());

    for (int node = 0; node < h_ref_row_offset.size() - 1; ++node)
    {
        SIZE_TYPE beg = h_ref_row_offset[node];
        SIZE_TYPE end = h_ref_row_offset[node + 1];
        for (SIZE_TYPE i = beg; i < end; ++i)
        {
            // h_ref_keys[i] != KEY_MAX && h_ref_keys[i] != KEY_NONE
            // KEY_TYPE mask = (KEY_TYPE)node << 32;
            // unsigned int dst = (h_ref_keys[i] - mask);
            unsigned int dst = (h_ref_keys[i] & 0xffffffff);
            if (h_ref_keys[i] != KEY_MAX && dst != COL_IDX_NONE && h_ref_values[i] != VALUE_NONE)
            {
                std::tuple<unsigned int, unsigned int, unsigned int> tup;
                unsigned int src = (h_ref_keys[i] >> 32);
                std::get<0>(tup) = src;
                std::get<1>(tup) = dst;
                std::get<2>(tup) = h_ref_values[i];
                vec.insert(tup);
            }
        }
    }

    return vec;
}

std::set<std::tuple<unsigned int, unsigned int, unsigned int>> get_reverse_csr_edge_list(GPMA &gpma)
{
    SIZE_TYPE* h_ref_row_offset = (SIZE_TYPE*) malloc((gpma.row_num+1) * sizeof(SIZE_TYPE));
    KEY_TYPE* h_ref_keys = (KEY_TYPE*) malloc((gpma.edge_count) * sizeof(KEY_TYPE));
    VALUE_TYPE* h_ref_values = (VALUE_TYPE*) malloc((gpma.edge_count) * sizeof(VALUE_TYPE));


    cErr(hipMemcpy(h_ref_row_offset, gpma.bwd_row_offset, (gpma.row_num+1) * sizeof(SIZE_TYPE), hipMemcpyDeviceToHost));
    cErr(hipMemcpy(h_ref_keys, gpma.bwd_keys, (gpma.edge_count) * sizeof(KEY_TYPE), hipMemcpyDeviceToHost));
    cErr(hipMemcpy(h_ref_values, gpma.bwd_values, (gpma.edge_count) * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost));

    std::set<std::tuple<unsigned int, unsigned int, unsigned int>> vec;

    // py::print("ROW_OFFSET: ", h_ref_row_offset.size());
    // py::print("COL_IDX_SIZE: ", h_ref_keys.size());
    // py::print("VALUES_SIZE: ", h_ref_values.size());

    for (int node = 0; node < gpma.row_num; ++node)
    {
        SIZE_TYPE beg = h_ref_row_offset[node];
        SIZE_TYPE end = h_ref_row_offset[node + 1];
        for (SIZE_TYPE i = beg; i < end; ++i)
        {
            // h_ref_keys[i] != KEY_MAX && h_ref_keys[i] != KEY_NONE
            // KEY_TYPE mask = (KEY_TYPE)node << 32;
            // unsigned int dst = (h_ref_keys[i] - mask);
            unsigned int dst = (h_ref_keys[i] & 0xffffffff);
            if (h_ref_keys[i] != KEY_MAX && dst != COL_IDX_NONE && h_ref_values[i] != VALUE_NONE)
            {
                std::tuple<unsigned int, unsigned int, unsigned int> tup;
                unsigned int src = (h_ref_keys[i] >> 32);
                std::get<0>(tup) = src;
                std::get<1>(tup) = dst;
                std::get<2>(tup) = h_ref_values[i];
                vec.insert(tup);
            }
        }
    }

    return vec;
}

PYBIND11_MODULE(gpma, m)
{
    m.doc() = "CPython module for GPMA"; // optional module docstring

    // GPMA APIs
    m.def("init_gpma", &init_gpma, "Initialises the CSR arrays using GPMA", py::arg("gpma"), py::arg("num_nodes"));
    m.def("init_graph_updates", &init_graph_updates, "Initialize graph updates", py::arg("gpma"), py::arg("updates"), py::arg("reverse_edges") = false);
    m.def("edge_update_t", &edge_update_t, "Performs the updates of the given timestamp", py::arg("gpma"), py::arg("timestamp"), py::arg("revert_update") = false);
    m.def("label_edges", &label_edges, "Labels edges of the given GPMA", py::arg("gpma"));
    m.def("build_backward_csr", &build_backward_csr, "Build the backward CSR for given GPMA", py::arg("gpma"));
    m.def("free_backward_csr", &free_backward_csr, "Free backward CSR resources from given GPMA", py::arg("gpma"));
    m.def("get_csr_ptrs", &get_csr_ptrs, "Returns the pointers to row_offset, col_indices and edge_ids", py::arg("gpma"), py::arg("is_backward") = false);
    m.def("get_in_degrees", &get_in_degrees, "Returns the in-degrees of all nodes in given GPMA", py::arg("gpma"));
    m.def("get_out_degrees", &get_out_degrees, "Returns the out-degrees of all nodes in given GPMA", py::arg("gpma"));
    
    // GPMA Logging APIs
    m.def("get_graph_attr", &get_graph_attr, "Returns the [num_nodes, num_edges]");
    m.def("print_gpma_info", &print_gpma_info, "Prints row_offset and col_indices for a given node");
    m.def("get_gpma_edge_list", &get_gpma_edge_list, "To get the edge list");
    m.def("get_reverse_csr_edge_list", &get_reverse_csr_edge_list, "To get the edge list of reverse");
    m.def("get_node_ids", &get_node_ids, "To get the node ids");

    py::class_<GPMA>(m, "GPMA")
        .def(py::init<>())
        .def("__copy__", [](const GPMA &self)
             { return GPMA(self); })
        .def(
            "__deepcopy__", [](const GPMA &self, py::dict)
            { return GPMA(self); },
            "memo"_a);
}

// Command used for compilation:
// /usr/local/cuda-11.7/bin/nvcc $(python3 -m pybind11 --includes) -shared -rdc=true --compiler-options '-fPIC'  -D__CDPRT_SUPPRESS_SYNC_DEPRECATION_WARNING -o gpma.so gpma.cu