#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <algorithm>
#include <queue>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <tuple>
#include <vector>
#include <string>
#include <cstring>
#include <iostream>

#include <chrono>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <hipcub/hipcub.hpp>
#include <thrust/system/cuda/experimental/pinned_allocator.h>

namespace py = pybind11;
using namespace pybind11::literals;
using namespace std;

using namespace std::chrono;

#define cErr(errcode)                             \
    {                                             \
        gpuAssert((errcode), __FILE__, __LINE__); \
    }
__inline__ __host__ __device__ void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

////////////////////////////////////////////////////////////////////////////

typedef struct _node
{
    // beginning and end of the associated region in the edge list
    uint32_t beginning;     // deleted = max int
    uint32_t end;           // end pointer is exclusive
    uint32_t num_neighbors; // number of edges with this node as source
    uint32_t in_degree;     // in-degree of a node -  number of edges going into the node

    _node(int beg = 0, int _end = 0, int num_neigh = 0, int in_deg = 0)
    {
        beginning = beg;
        end = _end;
        num_neighbors = num_neigh;
        in_degree = in_deg;
    }
} node_t;

typedef struct _edge
{
    uint32_t dest;  // destination of this edge in the graph, MAX_INT if this is a sentinel
    uint32_t value; // edge value of zero means it a null since we don't store 0 edges

    _edge(int _dest = 0, int _value = 0)
    {
        dest = _dest;
        value = _value;
    }
} edge_t;

typedef struct edge_list
{
    int N;
    int H;
    int logN;
    vector<edge_t> items;

    edge_list()
    {
        N = 0;
        H = 0;
        logN = 0;

        vector<edge_t> temp(0, 0);
        items = temp;
    }
} edge_list_t;

typedef struct _pair_int
{
    int x; // length in array
    int y; // depth

    _pair_int(int _x = 0, int _y = 0)
    {
        x = _x;
        y = _y;
    }
} pair_int;

typedef struct _pair_double
{
    double x;
    double y;

    _pair_double(double _x = 0, double _y = 0)
    {
        x = _x;
        y = _y;
    }
} pair_double;

////////////////////////////////////////////////////////////////////////////

static inline int bsf_word(int word)
{
    int result;
    __asm__ volatile("bsf %1, %0"
                     : "=r"(result)
                     : "r"(word));
    return result;
}

static inline int bsr_word(int word)
{
    int result;
    __asm__ volatile("bsr %1, %0"
                     : "=r"(result)
                     : "r"(word));
    return result;
}

// given index, return the starting index of the leaf it is in
int find_leaf(edge_list_t *list, int index)
{
    return (index / list->logN) * list->logN;
}

bool is_null(edge_t e) { return e.value == 0; }

bool is_sentinel(edge_t e)
{
    return e.dest == UINT32_MAX || e.value == UINT32_MAX;
}

// Possibly make this faster
uint32_t binary_search(edge_list_t *list, edge_t *elem, uint32_t start,
                       uint32_t end)
{
    while (start + 1 < end)
    {
        uint32_t mid = (start + end) / 2;

        edge_t item = list->items[mid];
        uint32_t change = 1;
        uint32_t check = mid;

        bool flag = true;
        while (is_null(item) && flag)
        {
            flag = false;
            check = mid + change;
            if (check < end)
            {
                flag = true;
                if (check <= end)
                {
                    item = list->items[check];
                    if (!is_null(item))
                    {
                        break;
                    }
                    else if (check == end)
                    {
                        break;
                    }
                }
            }
            check = mid - change;
            if (check >= start)
            {
                flag = true;
                item = list->items[check];
            }
            change++;
        }

        if (is_null(item) || start == check || end == check)
        {
            if (!is_null(item) && start == check && elem->dest <= item.dest)
            {
                return check;
            }
            return mid;
        }

        // if we found it, return
        if (elem->dest == item.dest)
        {
            return check;
        }
        else if (elem->dest < item.dest)
        {
            end =
                check; // if the searched for item is less than current item, set end
        }
        else
        {
            start = check;
            // otherwise, searched for item is more than current and we set start
        }
    }
    if (end < start)
    {
        start = end;
    }
    // handling the case where there is one element left
    // if you are leq, return start (index where elt is)
    // otherwise, return end (no element greater than you in the range)
    // printf("start = %d, end = %d, n = %d\n", start,end, list->N);
    if (elem->dest <= list->items[start].dest && !is_null(list->items[start]))
    {
        return start;
    }
    return end;
}

// get density of a node
double get_density(edge_list_t *list, int index, int len)
{
    int full = 0;
    for (int i = index; i < index + len; i++)
    {
        full += (!is_null(list->items[i]));
    }
    double full_d = (double)full;
    return full_d / len;
}

int find_node(int index, int len) { return (index / len) * len; }

pair_double density_bound(edge_list_t *list, int depth)
{
    pair_double pair;

    // between 1/4 and 1/2
    // pair.x = 1.0/2.0 - (( .25*depth)/list->H);
    // between 1/8 and 1/4
    pair.x = 1.0 / 4.0 - ((.125 * depth) / list->H);
    pair.y = 3.0 / 4.0 + ((.25 * depth) / list->H);
    return pair;
}

bool edge_equals(edge_t e1, edge_t e2)
{
    return e1.dest == e2.dest && e1.value == e2.value;
}

uint32_t find_elem_pointer(edge_list_t *list, uint32_t index, edge_t elem)
{
    edge_t item = list->items[index];
    while (!edge_equals(item, elem))
    {
        item = list->items[++index];
    }
    return index;
}

////////////////////////////////////////////////////////////////////////////

class PCSR
{
public:
    // data members
    std::vector<node_t> nodes;
    std::vector<uint32_t> in_degrees;
    std::vector<uint32_t> out_degrees;

    edge_list_t edges;
    uint32_t edge_count;

    // replacing device vectors
    uint32_t *row_offset_pinned;
    uint32_t *column_indices_pinned;
    uint32_t *eids_pinned;
    uint32_t *node_ids_pinned;

    // replacing device vectors
    uint32_t *row_offset_device;
    uint32_t *column_indices_device;
    uint32_t *eids_device;
    uint32_t *node_ids_device;

    // member functions
    PCSR(uint32_t init_n, uint32_t max_edge_count);
    // ~PCSR();
    
    // PCSR specific internal functions
    uint64_t get_n();
    void add_node();
    uint32_t insert(uint32_t index, edge_t elem, uint32_t src);
    void double_list();
    int slide_right(int index);
    void slide_left(int index);
    void fix_sentinel(int32_t node_index, int in);
    void redistribute(int index, int len);
    void add_edge(uint32_t src, uint32_t dest, uint32_t value);
    void add_edge_update(uint32_t src, uint32_t dest, uint32_t value);
    void delete_edge(uint32_t src, uint32_t dest);

    // exposed APIs
    void edge_update_list(std::vector<std::tuple<uint32_t, uint32_t>> edge_list, bool is_delete, bool is_reverse_edge);
    void label_edges();
    float build_csr();
    float build_reverse_csr();
    std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t, std::uintptr_t> get_csr_ptrs();
    vector<tuple<uint32_t, uint32_t, uint32_t>> get_edges();
    void move_pinned_to_gpu();
};

////////////////////////////////////////////////////////////////////////////

PCSR::PCSR(uint32_t init_n, uint32_t max_num_edges)
{
    if (init_n != 0)
    {
        edges.N = 2 << bsr_word(init_n);
        edges.logN = (1 << bsr_word(bsr_word(edges.N) + 1));
        edges.H = bsr_word(edges.N / edges.logN);

        edges.items.resize(edges.N);
        edge_count = 0;

        for (int i = 0; i < edges.N; i++)
        {
            edge_t new_edge(0, 0);
            edges.items[i] = new_edge;
        }

        for (int i = 0; i < init_n; i++)
        {
            add_node();
        }

        in_degrees.resize(init_n);
        out_degrees.resize(init_n);

        cErr(hipHostMalloc(&row_offset_pinned, sizeof(uint32_t) * (init_n + 1)));
        cErr(hipHostMalloc(&column_indices_pinned, sizeof(uint32_t) * max_num_edges));
        cErr(hipHostMalloc(&eids_pinned, sizeof(uint32_t) * max_num_edges));
        cErr(hipHostMalloc(&node_ids_pinned, sizeof(uint32_t) * init_n));

        cErr(hipMalloc(&row_offset_device, sizeof(uint32_t) * (init_n + 1)));
        cErr(hipMalloc(&column_indices_device, sizeof(uint32_t) * max_num_edges));
        cErr(hipMalloc(&eids_device, sizeof(uint32_t) * max_num_edges));
        cErr(hipMalloc(&node_ids_device, sizeof(uint32_t) * init_n));
    }
}

// PCSR::~PCSR()
// {
//     cErr(hipFree(row_offset_pinned));
//     cErr(hipFree(column_indices_pinned));
//     cErr(hipFree(eids_pinned));

//     cErr(hipFree(row_offset_device));
//     cErr(hipFree(column_indices_device));
//     cErr(hipFree(eids_device));

//     nodes.clear();
//     in_degrees.clear();
//     out_degrees.clear();
// }

// add a node to the graph
void PCSR::add_node()
{
    node_t node;
    int len = nodes.size();

    edge_t sentinel;
    sentinel.dest = UINT32_MAX; // placeholder
    sentinel.value = len;       // back pointer

    if (len > 0)
    {
        node.beginning = nodes[len - 1].end;
        node.end = node.beginning + 1;
    }
    else
    {
        node.beginning = 0;
        node.end = 1;
        sentinel.value = UINT32_MAX;
    }
    node.num_neighbors = 0;

    nodes.push_back(node);
    insert(node.beginning, sentinel, nodes.size() - 1);
}

uint32_t PCSR::insert(uint32_t index, edge_t elem, uint32_t src)
{
    int node_index = find_leaf(&edges, index);
    int level = edges.H;
    int len = edges.logN;

    // always deposit on the left
    if (is_null(edges.items[index]))
    {
        edges.items[index].value = elem.value;
        edges.items[index].dest = elem.dest;
    }
    else
    {
        // if the edge already exists in the graph, update its value
        // do not make another edge
        // return index of the edge that already exists
        if (!is_sentinel(elem) && edges.items[index].dest == elem.dest)
        {
            edges.items[index].value = elem.value;
            return index;
        }
        if (index == edges.N - 1)
        {
            // when adding to the end double then add edge
            double_list();
            node_t node = nodes[src];
            uint32_t loc_to_add =
                binary_search(&edges, &elem, node.beginning + 1, node.end);
            return insert(loc_to_add, elem, src);
        }
        else
        {
            if (slide_right(index) == -1)
            {
                index -= 1;
                slide_left(index);
            }
        }
        edges.items[index].value = elem.value;
        edges.items[index].dest = elem.dest;
    }

    double density = get_density(&edges, node_index, len);

    // spill over into next level up, node is completely full.
    if (density == 1)
    {
        node_index = find_node(node_index, len * 2);
        redistribute(node_index, len * 2);
    }
    else
    {
        // makes the last slot in a section empty so you can always slide right
        redistribute(node_index, len);
    }

    // get density of the leaf you are in
    pair_double density_b = density_bound(&edges, level);
    density = get_density(&edges, node_index, len);

    // while density too high, go up the implicit tree
    // go up to the biggest node above the density bound
    while (density >= density_b.y)
    {
        len *= 2;
        if (len <= edges.N)
        {
            level--;
            node_index = find_node(node_index, len);
            density_b = density_bound(&edges, level);
            density = get_density(&edges, node_index, len);
        }
        else
        {
            // if you reach the root, double the list
            double_list();

            // search from the beginning because list was doubled
            return find_elem_pointer(&edges, 0, elem);
        }
    }
    redistribute(node_index, len);

    return find_elem_pointer(&edges, node_index, elem);
}

void PCSR::double_list()
{
    edges.N *= 2;
    edges.logN = (1 << bsr_word(bsr_word(edges.N) + 1));
    edges.H = bsr_word(edges.N / edges.logN);

    edges.items.resize(edges.N);
    for (int i = edges.N / 2; i < edges.N; i++)
    {
        edge_t new_edge(0, 0);
        edges.items[i] = new_edge;
    }

    redistribute(0, edges.N);
}

int PCSR::slide_right(int index)
{
    int rval = 0;
    edge_t el = edges.items[index];
    edges.items[index].dest = 0;
    edges.items[index].value = 0;
    index++;
    while (index < edges.N && !is_null(edges.items[index]))
    {
        edge_t temp = edges.items[index];
        edges.items[index] = el;
        if (!is_null(el) && is_sentinel(el))
        {
            // fixing pointer of node that goes to this sentinel
            uint32_t node_index = el.value;
            if (node_index == UINT32_MAX)
            {
                node_index = 0;
            }
            fix_sentinel(node_index, index);
        }
        el = temp;
        index++;
    }
    if (!is_null(el) && is_sentinel(el))
    {
        // fixing pointer of node that goes to this sentinel
        uint32_t node_index = el.value;
        if (node_index == UINT32_MAX)
        {
            node_index = 0;
        }
        fix_sentinel(node_index, index);
    }
    // TODO There might be an issue with this going of the end sometimes
    if (index == edges.N)
    {
        index--;
        slide_left(index);
        rval = -1;
        printf("slide off the end on the right, should be rare\n");
    }
    edges.items[index] = el;
    return rval;
}

void PCSR::slide_left(int index)
{
    edge_t el = edges.items[index];
    edges.items[index].dest = 0;
    edges.items[index].value = 0;

    index--;
    while (index >= 0 && !is_null(edges.items[index]))
    {
        edge_t temp = edges.items[index];
        edges.items[index] = el;
        if (!is_null(el) && is_sentinel(el))
        {
            // fixing pointer of node that goes to this sentinel
            uint32_t node_index = el.value;
            if (node_index == UINT32_MAX)
            {
                node_index = 0;
            }

            fix_sentinel(node_index, index);
        }
        el = temp;
        index--;
    }

    if (index == -1)
    {
        double_list();

        slide_right(0);
        index = 0;
    }
    if (!is_null(el) && is_sentinel(el))
    {
        // fixing pointer of node that goes to this sentinel
        uint32_t node_index = el.value;
        if (node_index == UINT32_MAX)
        {
            node_index = 0;
        }
        fix_sentinel(node_index, index);
    }

    edges.items[index] = el;
}

void PCSR::fix_sentinel(int32_t node_index, int in)
{
    nodes[node_index].beginning = in;
    if (node_index > 0)
    {
        nodes[node_index - 1].end = in;
    }
    if (node_index == nodes.size() - 1)
    {
        nodes[node_index].end = edges.N - 1;
    }
}

void PCSR::redistribute(int index, int len)
{
    // printf("REDISTRIBUTE: \n");
    // print_array();
    // std::vector<edge_t> space(len); //
    edge_t new_edge;
    vector<edge_t> space(len, new_edge);

    int j = 0;

    // move all items in ofm in the range into
    // a temp array
    for (int i = index; i < index + len; i++)
    {
        space[j] = edges.items[i];
        // counting non-null edges
        j += (!is_null(edges.items[i]));
        // setting section to null
        edges.items[i].value = 0;
        edges.items[i].dest = 0;
    }

    // evenly redistribute for a uniform density
    double index_d = index;
    double step = ((double)len) / j;
    for (int i = 0; i < j; i++)
    {
        int in = index_d;

        edges.items[in] = space[i];
        if (is_sentinel(space[i]))
        {
            // fixing pointer of node that goes to this sentinel
            uint32_t node_index = space[i].value;
            if (node_index == UINT32_MAX)
            {
                node_index = 0;
            }
            fix_sentinel(node_index, in);
        }
        index_d += step;
    }
    //   free(space);
}

void PCSR::add_edge(uint32_t src, uint32_t dest, uint32_t value)
{
    // cout << "Adding edge (" << src << "," << dest << ")\n";
    if (value != 0)
    {
        node_t node = nodes[src];
        nodes[src].num_neighbors++;
        nodes[dest].in_degree++;

        edge_t e;
        e.dest = dest;
        e.value = value;

        uint32_t loc_to_add =
            binary_search(&edges, &e, node.beginning + 1, node.end);
        insert(loc_to_add, e, src);
        ++edge_count;
    }
}

void PCSR::add_edge_update(uint32_t src, uint32_t dest, uint32_t value)
{
    if (value != 0)
    {
        node_t node = nodes[src];

        edge_t e;
        e.dest = dest;
        e.value = value;

        uint32_t loc_to_add =
            binary_search(&edges, &e, node.beginning + 1, node.end);
        if (edges.items[loc_to_add].dest == dest)
        {
            edges.items[loc_to_add].value = value;
            return;
        }
        nodes[src].num_neighbors++;
        nodes[dest].in_degree++;
        insert(loc_to_add, e, src);
        ++edge_count;
    }
}

void PCSR::delete_edge(uint32_t src, uint32_t dest)
{
    // cout << "Deleting edge (" << src << "," << dest << ")\n";
    edge_t e;
    e.value = 0;
    e.dest = dest;
    uint32_t loc = binary_search(&edges, &e, nodes[src].beginning + 1, nodes[src].end);

    if (!is_null(edges.items[loc]) && edges.items[loc].dest == dest)
    {
        edges.items[loc].value = 0;
        nodes[src].num_neighbors -= 1;
        nodes[dest].in_degree -= 1;
        --edge_count;
    }
}

uint64_t PCSR::get_n()
{
    return nodes.size();
}

vector<tuple<uint32_t, uint32_t, uint32_t>> PCSR::get_edges()
{
    uint64_t n = get_n();
    vector<tuple<uint32_t, uint32_t, uint32_t>> output;

    output.resize(edge_count);
    int iter = 0;
    for (int i = 0; i < n; i++)
    {
        uint32_t start = nodes[i].beginning;
        uint32_t end = nodes[i].end;
        for (int j = start + 1; j < end; j++)
        {
            if (!is_null(edges.items[j]))
            {
                output[iter] =
                    make_tuple(i, edges.items[j].dest, edges.items[j].value);
                iter += 1;
            }
        }
    }
    return output;
}

void PCSR::label_edges()
{
    uint32_t column_indices_size = edges.items.size();
    uint32_t counter = 1;
    for (int i = 0; i < column_indices_size; ++i)
    {
        if (!is_sentinel(edges.items[i]) && !is_null(edges.items[i]))
        {
            edges.items[i].value = counter;
            ++counter;
        }
    }
}

void PCSR::edge_update_list(std::vector<std::tuple<uint32_t, uint32_t>> edge_list, bool is_delete = false, bool is_reverse_edge = false)
{
    bool is_reverse_edge_local = is_reverse_edge;
    bool is_delete_local = is_delete;

    for (auto &edge : edge_list)
    {
        uint32_t src = (is_reverse_edge_local == true) ? std::get<1>(edge) : std::get<0>(edge);
        uint32_t dst = (is_reverse_edge_local == true) ? std::get<0>(edge) : std::get<1>(edge);

        if (is_delete_local){
            in_degrees[dst] -= 1;
            out_degrees[src] -= 1;
            delete_edge(src, dst);
        }else{
            in_degrees[dst] += 1;
            out_degrees[src] += 1;
            add_edge(src, dst, 1);
        }
    }
}

float PCSR::build_reverse_csr()
{
    uint64_t n = get_n();
    // computing the bwd row offsets
    row_offset_pinned[0] = in_degrees[0];
    for(int i=1; i<in_degrees.size(); ++i){
        row_offset_pinned[i] = row_offset_pinned[i-1] + in_degrees[i];
    }
    row_offset_pinned[in_degrees.size()] = edge_count;
    
    for (int i = 0; i < n; i++)
    {
        uint32_t start = nodes[i].beginning;
        uint32_t end = nodes[i].end;
        for (int j = start + 1; j < end; j++)
        {
            if (!is_sentinel(edges.items[j]) && !is_null(edges.items[j]))
            {
                row_offset_pinned[edges.items[j].dest] -= 1;

                int col_index = row_offset_pinned[edges.items[j].dest];
                column_indices_pinned[col_index] = i;
                eids_pinned[col_index] = edges.items[j].value;
            }
        }
    }

    // Obtaining the sorted order of node ids (in descending order of in-degrees)
    std::vector<std::pair<uint32_t, uint32_t>> degree_id_pairs = std::vector<std::pair<uint32_t, uint32_t>>();
    for(int i=0; i<in_degrees.size(); ++i){
        degree_id_pairs.push_back(std::make_pair(in_degrees[i],i));
    }

    std::sort(degree_id_pairs.begin(), degree_id_pairs.end(), [ ](const std::pair<int, int>& lhs, const std::pair<int, int>& rhs ) {
        return lhs.first > rhs.first;
    });

    for(int i=0; i<degree_id_pairs.size(); ++i){
        node_ids_pinned[i] = degree_id_pairs[i].second;
    }

    auto start_time = std::chrono::high_resolution_clock::now();
    move_pinned_to_gpu();
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> move_to_gpu_time = (end_time - start_time);
    return move_to_gpu_time.count();
}

float PCSR::build_csr()
{
    // computing the bwd row offsets
    uint64_t n = get_n();
    row_offset_pinned[0] = out_degrees[0];
    for(int i=1; i<out_degrees.size(); ++i){
        row_offset_pinned[i] = row_offset_pinned[i-1] + out_degrees[i];
    }
    row_offset_pinned[out_degrees.size()] = edge_count;
    
    for (int i = 0; i < n; i++)
    {
        uint32_t start = nodes[i].beginning;
        uint32_t end = nodes[i].end;
        for (int j = start + 1; j < end; j++)
        {
            if (!is_sentinel(edges.items[j]) && !is_null(edges.items[j]))
            {
                row_offset_pinned[i] -= 1;
                column_indices_pinned[row_offset_pinned[i]] = edges.items[j].dest;
                eids_pinned[row_offset_pinned[i]] = edges.items[j].value;
            }
        }
    }

    // Obtaining the sorted order of node ids (in descending order of out-degrees)
    std::vector<std::pair<uint32_t, uint32_t>> degree_id_pairs = std::vector<std::pair<uint32_t, uint32_t>>();
    for(int i=0; i<out_degrees.size(); ++i){
        degree_id_pairs.push_back(std::make_pair(out_degrees[i],i));
    }

    std::sort(degree_id_pairs.begin(), degree_id_pairs.end(), [ ](const std::pair<int, int>& lhs, const std::pair<int, int>& rhs ) {
        return lhs.first > rhs.first;
    });

    for(int i=0; i<degree_id_pairs.size(); ++i){
        node_ids_pinned[i] = degree_id_pairs[i].second;
    }

    auto start_time = std::chrono::high_resolution_clock::now();
    move_pinned_to_gpu();
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> move_to_gpu_time = (end_time - start_time);
    return move_to_gpu_time.count();
}

void PCSR::move_pinned_to_gpu(){
    cErr(hipMemcpy(row_offset_device, row_offset_pinned, sizeof(uint32_t) * (get_n() + 1), hipMemcpyHostToDevice));
    cErr(hipMemcpy(column_indices_device, column_indices_pinned, sizeof(uint32_t) * edge_count, hipMemcpyHostToDevice));
    cErr(hipMemcpy(eids_device, eids_pinned, sizeof(uint32_t) * edge_count, hipMemcpyHostToDevice));
    cErr(hipMemcpy(node_ids_device, node_ids_pinned, sizeof(uint32_t) * get_n(), hipMemcpyHostToDevice));
}

std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t, std::uintptr_t> PCSR::get_csr_ptrs(){
    std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t, std::uintptr_t> t;
    std::get<0>(t) = (std::uintptr_t)row_offset_device;
    std::get<1>(t) = (std::uintptr_t)column_indices_device;
    std::get<2>(t) = (std::uintptr_t)eids_device;
    std::get<3>(t) = (std::uintptr_t)node_ids_device;
    return t;
}

std::vector<std::vector<uint32_t>> read_gpu_csr(PCSR &pcsr){
    std::vector<uint32_t> row_offset(pcsr.get_n() + 1);
    std::vector<uint32_t> column_indices(pcsr.edge_count);
    std::vector<uint32_t> eids(pcsr.edge_count);
    std::vector<uint32_t> node_ids(pcsr.get_n());

    cErr(hipMemcpy(row_offset.data(), pcsr.row_offset_device, sizeof(uint32_t) * (pcsr.get_n() + 1), hipMemcpyDeviceToHost));
    cErr(hipMemcpy(column_indices.data(), pcsr.column_indices_device, sizeof(uint32_t) * pcsr.edge_count, hipMemcpyDeviceToHost));
    cErr(hipMemcpy(eids.data(), pcsr.eids_device, sizeof(uint32_t) * pcsr.edge_count, hipMemcpyDeviceToHost));
    cErr(hipMemcpy(node_ids.data(), pcsr.node_ids_device, sizeof(uint32_t) * pcsr.get_n(), hipMemcpyDeviceToHost));

    std::vector<std::vector<uint32_t>> res;
    res.push_back(row_offset);
    res.push_back(column_indices);
    res.push_back(eids);
    res.push_back(node_ids);
    return res;
}

////////////////////////////////////////////////////////////////////////////

// PCSR Python Module

PYBIND11_MODULE(pcsr, m)
{
    m.doc() = "PCSR Dynamic Graph Representation";
    m.def("read_gpu_csr", &read_gpu_csr, "Read CSR arrays from the GPU");

    py::class_<PCSR>(m, "PCSR")
        .def(py::init<int, int>(), py::arg("init_n"), py::arg("max_edge_count"))
        .def_readwrite("in_degrees", &PCSR::in_degrees)
        .def_readwrite("out_degrees", &PCSR::out_degrees)
        .def_readwrite("edge_count", &PCSR::edge_count)
        .def("get_n", &PCSR::get_n)
        .def("edge_update_list", &PCSR::edge_update_list, py::arg("edge_list"), py::arg("is_delete") = false, py::arg("is_reverse_edge") = false)
        .def("label_edges", &PCSR::label_edges, "Creates edge labels for the current GPMA")
        .def("get_edges", &PCSR::get_edges)
        .def("build_csr", &PCSR::build_csr)
        .def("build_reverse_csr", &PCSR::build_reverse_csr)
        .def("get_csr_ptrs", &PCSR::get_csr_ptrs)
        .def("__copy__", [](const PCSR &self)
             { return PCSR(self); })
        .def(
            "__deepcopy__", [](const PCSR &self, py::dict)
            { return PCSR(self); },
            "memo"_a);
}

// /usr/local/cuda-11.7/bin/nvcc $(python3 -m pybind11 --includes) -shared -rdc=true --compiler-options '-fPIC'  -D__CDPRT_SUPPRESS_SYNC_DEPRECATION_WARNING -o pcsr.so pcsr.cu
