#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <algorithm>
#include <queue>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <tuple>
#include <vector>
#include <string>
#include <cstring>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <hipcub/hipcub.hpp>

namespace py = pybind11;
using namespace pybind11::literals;
using namespace std;

typedef thrust::device_vector<int> DEV_VEC;
#define RAW_PTR(x) thrust::raw_pointer_cast((x).data())

////////////////////////////////////////////////////////////////////////////

typedef struct _node
{
    // beginning and end of the associated region in the edge list
    uint32_t beginning;     // deleted = max int
    uint32_t end;           // end pointer is exclusive
    uint32_t num_neighbors; // number of edges with this node as source
    uint32_t in_degree;     // in-degree of a node -  number of edges going into the node

    _node(int beg = 0, int _end = 0, int num_neigh = 0, int in_deg = 0)
    {
        beginning = beg;
        end = _end;
        num_neighbors = num_neigh;
        in_degree = in_deg;
    }
} node_t;

typedef struct _edge
{
    uint32_t dest;  // destination of this edge in the graph, MAX_INT if this is a sentinel
    uint32_t value; // edge value of zero means it a null since we don't store 0 edges

    _edge(int _dest = 0, int _value = 0)
    {
        dest = _dest;
        value = _value;
    }
} edge_t;

typedef struct edge_list
{
    int N;
    int H;
    int logN;
    vector<edge_t> items;

    edge_list()
    {
        N = 0;
        H = 0;
        logN = 0;

        vector<edge_t> temp(0, 0);
        items = temp;
    }
} edge_list_t;

typedef struct _pair_int
{
    int x; // length in array
    int y; // depth

    _pair_int(int _x = 0, int _y = 0)
    {
        x = _x;
        y = _y;
    }
} pair_int;

typedef struct _pair_double
{
    double x;
    double y;

    _pair_double(double _x = 0, double _y = 0)
    {
        x = _x;
        y = _y;
    }
} pair_double;

////////////////////////////////////////////////////////////////////////////

static inline int bsf_word(int word)
{
    int result;
    __asm__ volatile("bsf %1, %0"
                     : "=r"(result)
                     : "r"(word));
    return result;
}

static inline int bsr_word(int word)
{
    int result;
    __asm__ volatile("bsr %1, %0"
                     : "=r"(result)
                     : "r"(word));
    return result;
}

// given index, return the starting index of the leaf it is in
int find_leaf(edge_list_t *list, int index)
{
    return (index / list->logN) * list->logN;
}

bool is_null(edge_t e) { return e.value == 0; }

bool is_sentinel(edge_t e)
{
    return e.dest == UINT32_MAX || e.value == UINT32_MAX;
}

uint32_t binary_search(edge_list_t *list, edge_t *elem, uint32_t start,
                       uint32_t end)
{
    while (start + 1 < end)
    {
        uint32_t mid = (start + end) / 2;

        edge_t item = list->items[mid];
        uint32_t change = 1;
        uint32_t check = mid;

        bool flag = true;
        while (is_null(item) && flag)
        {
            flag = false;
            check = mid + change;
            if (check < end)
            {
                flag = true;
                if (check <= end)
                {
                    item = list->items[check];
                    if (!is_null(item))
                    {
                        break;
                    }
                    else if (check == end)
                    {
                        break;
                    }
                }
            }
            check = mid - change;
            if (check >= start)
            {
                flag = true;
                item = list->items[check];
            }
            change++;
        }

        if (is_null(item) || start == check || end == check)
        {
            if (!is_null(item) && start == check && elem->dest <= item.dest)
            {
                return check;
            }
            return mid;
        }

        // if we found it, return
        if (elem->dest == item.dest)
        {
            return check;
        }
        else if (elem->dest < item.dest)
        {
            end =
                check; // if the searched for item is less than current item, set end
        }
        else
        {
            start = check;
            // otherwise, searched for item is more than current and we set start
        }
    }
    if (end < start)
    {
        start = end;
    }
    // handling the case where there is one element left
    // if you are leq, return start (index where elt is)
    // otherwise, return end (no element greater than you in the range)
    // printf("start = %d, end = %d, n = %d\n", start,end, list->N);
    if (elem->dest <= list->items[start].dest && !is_null(list->items[start]))
    {
        return start;
    }
    return end;
}

// get density of a node
double get_density(edge_list_t *list, int index, int len)
{
    int full = 0;
    for (int i = index; i < index + len; i++)
    {
        full += (!is_null(list->items[i]));
    }
    double full_d = (double)full;
    return full_d / len;
}

int find_node(int index, int len) { return (index / len) * len; }

pair_double density_bound(edge_list_t *list, int depth)
{
    pair_double pair;

    // between 1/4 and 1/2
    // pair.x = 1.0/2.0 - (( .25*depth)/list->H);
    // between 1/8 and 1/4
    pair.x = 1.0 / 4.0 - ((.125 * depth) / list->H);
    pair.y = 3.0 / 4.0 + ((.25 * depth) / list->H);
    return pair;
}

bool edge_equals(edge_t e1, edge_t e2)
{
    return e1.dest == e2.dest && e1.value == e2.value;
}

uint32_t find_elem_pointer(edge_list_t *list, uint32_t index, edge_t elem)
{
    edge_t item = list->items[index];
    while (!edge_equals(item, elem))
    {
        item = list->items[++index];
    }
    return index;
}

////////////////////////////////////////////////////////////////////////////

class PCSR
{
public:
    // data members
    std::vector<node_t> nodes;
    edge_list_t edges;
    uint32_t edge_count;

    DEV_VEC row_offset_device;
    DEV_VEC column_indices_device;
    DEV_VEC eids_device;

    // member functions
    PCSR(uint32_t init_n);
    void init_graph(std::string graph_file_path);

    void add_node();
    void add_edge(uint32_t src, uint32_t dest, uint32_t value);
    void add_edge_update(uint32_t src, uint32_t dest, uint32_t value);
    void edge_update_list(std::vector<std::tuple<uint32_t, uint32_t>> edge_list, bool is_delete, bool is_reverse_edge);
    void delete_edge(uint32_t src, uint32_t dest);
    uint64_t get_n();
    // vector<tuple<uint32_t, uint32_t, uint32_t>> get_edges();
    void print_graph();
    void print_array();
    std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t> get_csr_ptrs(std::vector<int> eids);
    // void label_edges();
    // uint32_t find_edge_id(uint32_t src, uint32_t dest);
    // std::tuple<uint32_t, uint32_t> get_graph_attr();

    uint32_t insert(uint32_t index, edge_t elem, uint32_t src);
    void double_list();
    int slide_right(int index);
    void slide_left(int index);
    void fix_sentinel(int32_t node_index, int in);
    void redistribute(int index, int len);
};

////////////////////////////////////////////////////////////////////////////

// removed default argument of 0
PCSR::PCSR(uint32_t init_n)
{
    if (init_n != 0)
    {
        edges.N = 2 << bsr_word(init_n);
        edges.logN = (1 << bsr_word(bsr_word(edges.N) + 1));
        edges.H = bsr_word(edges.N / edges.logN);

        edges.items.resize(edges.N);
        edge_count = 0;

        for (int i = 0; i < edges.N; i++)
        {
            edge_t new_edge(0, 0);
            edges.items[i] = new_edge;
        }

        for (int i = 0; i < init_n; i++)
        {
            add_node();
        }
    }
}

void PCSR::init_graph(std::string graph_file_path)
{

    FILE *fp;
    fp = fopen(graph_file_path.c_str(), "r");
    if (!fp)
    {
        printf("Open file failed.\n");
        exit(-1);
    }

    // loading the number of nodes and edges present in
    // the graph into the variables node_size and edge_size respectively
    int node_size, edge_size;
    fscanf(fp, "%d %d", &node_size, &edge_size);

    edges.N = 2 << bsr_word(node_size);
    edges.logN = (1 << bsr_word(bsr_word(edges.N) + 1));
    edges.H = bsr_word(edges.N / edges.logN);
    edge_count = 0;

    edges.items.resize(edges.N);
    for (int i = 0; i < edges.N; i++)
    {
        edge_t new_edge(0, 0);
        edges.items[i] = new_edge;
    }

    for (int i = 0; i < node_size; i++)
    {
        add_node();
    }

    for (int i = 0; i < edge_size; i++)
    {
        int source, dest, edge_value;
        fscanf(fp, "%d %d %d", &source, &dest, &edge_value);
        add_edge(source, dest, edge_value);
    }
}

// add a node to the graph
void PCSR::add_node()
{
    node_t node;
    int len = nodes.size();

    edge_t sentinel;
    sentinel.dest = UINT32_MAX; // placeholder
    sentinel.value = len;       // back pointer

    if (len > 0)
    {
        node.beginning = nodes[len - 1].end;
        node.end = node.beginning + 1;
    }
    else
    {
        node.beginning = 0;
        node.end = 1;
        sentinel.value = UINT32_MAX;
    }
    node.num_neighbors = 0;

    nodes.push_back(node);
    insert(node.beginning, sentinel, nodes.size() - 1);
}

uint32_t PCSR::insert(uint32_t index, edge_t elem, uint32_t src)
{
    int node_index = find_leaf(&edges, index);
    int level = edges.H;
    int len = edges.logN;

    // always deposit on the left
    if (is_null(edges.items[index]))
    {
        edges.items[index].value = elem.value;
        edges.items[index].dest = elem.dest;
    }
    else
    {
        // if the edge already exists in the graph, update its value
        // do not make another edge
        // return index of the edge that already exists
        if (!is_sentinel(elem) && edges.items[index].dest == elem.dest)
        {
            edges.items[index].value = elem.value;
            return index;
        }
        if (index == edges.N - 1)
        {
            // when adding to the end double then add edge
            double_list();
            node_t node = nodes[src];
            uint32_t loc_to_add =
                binary_search(&edges, &elem, node.beginning + 1, node.end);
            return insert(loc_to_add, elem, src);
        }
        else
        {
            if (slide_right(index) == -1)
            {
                index -= 1;
                slide_left(index);
            }
        }
        edges.items[index].value = elem.value;
        edges.items[index].dest = elem.dest;
    }

    double density = get_density(&edges, node_index, len);

    // spill over into next level up, node is completely full.
    if (density == 1)
    {
        node_index = find_node(node_index, len * 2);
        redistribute(node_index, len * 2);
    }
    else
    {
        // makes the last slot in a section empty so you can always slide right
        redistribute(node_index, len);
    }

    // get density of the leaf you are in
    pair_double density_b = density_bound(&edges, level);
    density = get_density(&edges, node_index, len);

    // while density too high, go up the implicit tree
    // go up to the biggest node above the density bound
    while (density >= density_b.y)
    {
        len *= 2;
        if (len <= edges.N)
        {
            level--;
            node_index = find_node(node_index, len);
            density_b = density_bound(&edges, level);
            density = get_density(&edges, node_index, len);
        }
        else
        {
            // if you reach the root, double the list
            double_list();

            // search from the beginning because list was doubled
            return find_elem_pointer(&edges, 0, elem);
        }
    }
    redistribute(node_index, len);

    return find_elem_pointer(&edges, node_index, elem);
}

void PCSR::double_list()
{
    edges.N *= 2;
    edges.logN = (1 << bsr_word(bsr_word(edges.N) + 1));
    edges.H = bsr_word(edges.N / edges.logN);

    edges.items.resize(edges.N);
    for (int i = edges.N / 2; i < edges.N; i++)
    {
        edge_t new_edge(0, 0);
        edges.items[i] = new_edge;
    }

    redistribute(0, edges.N);
}

int PCSR::slide_right(int index)
{
    int rval = 0;
    edge_t el = edges.items[index];
    edges.items[index].dest = 0;
    edges.items[index].value = 0;
    index++;
    while (index < edges.N && !is_null(edges.items[index]))
    {
        edge_t temp = edges.items[index];
        edges.items[index] = el;
        if (!is_null(el) && is_sentinel(el))
        {
            // fixing pointer of node that goes to this sentinel
            uint32_t node_index = el.value;
            if (node_index == UINT32_MAX)
            {
                node_index = 0;
            }
            fix_sentinel(node_index, index);
        }
        el = temp;
        index++;
    }
    if (!is_null(el) && is_sentinel(el))
    {
        // fixing pointer of node that goes to this sentinel
        uint32_t node_index = el.value;
        if (node_index == UINT32_MAX)
        {
            node_index = 0;
        }
        fix_sentinel(node_index, index);
    }
    // TODO There might be an issue with this going of the end sometimes
    if (index == edges.N)
    {
        index--;
        slide_left(index);
        rval = -1;
        printf("slide off the end on the right, should be rare\n");
    }
    edges.items[index] = el;
    return rval;
}

void PCSR::slide_left(int index)
{
    edge_t el = edges.items[index];
    edges.items[index].dest = 0;
    edges.items[index].value = 0;

    index--;
    while (index >= 0 && !is_null(edges.items[index]))
    {
        edge_t temp = edges.items[index];
        edges.items[index] = el;
        if (!is_null(el) && is_sentinel(el))
        {
            // fixing pointer of node that goes to this sentinel
            uint32_t node_index = el.value;
            if (node_index == UINT32_MAX)
            {
                node_index = 0;
            }

            fix_sentinel(node_index, index);
        }
        el = temp;
        index--;
    }

    if (index == -1)
    {
        double_list();

        slide_right(0);
        index = 0;
    }
    if (!is_null(el) && is_sentinel(el))
    {
        // fixing pointer of node that goes to this sentinel
        uint32_t node_index = el.value;
        if (node_index == UINT32_MAX)
        {
            node_index = 0;
        }
        fix_sentinel(node_index, index);
    }

    edges.items[index] = el;
}

void PCSR::fix_sentinel(int32_t node_index, int in)
{
    nodes[node_index].beginning = in;
    if (node_index > 0)
    {
        nodes[node_index - 1].end = in;
    }
    if (node_index == nodes.size() - 1)
    {
        nodes[node_index].end = edges.N - 1;
    }
}

void PCSR::redistribute(int index, int len)
{
    // printf("REDISTRIBUTE: \n");
    // print_array();
    // std::vector<edge_t> space(len); //
    edge_t new_edge;
    vector<edge_t> space(len, new_edge);

    int j = 0;

    // move all items in ofm in the range into
    // a temp array
    for (int i = index; i < index + len; i++)
    {
        space[j] = edges.items[i];
        // counting non-null edges
        j += (!is_null(edges.items[i]));
        // setting section to null
        edges.items[i].value = 0;
        edges.items[i].dest = 0;
    }

    // evenly redistribute for a uniform density
    double index_d = index;
    double step = ((double)len) / j;
    for (int i = 0; i < j; i++)
    {
        int in = index_d;

        edges.items[in] = space[i];
        if (is_sentinel(space[i]))
        {
            // fixing pointer of node that goes to this sentinel
            uint32_t node_index = space[i].value;
            if (node_index == UINT32_MAX)
            {
                node_index = 0;
            }
            fix_sentinel(node_index, in);
        }
        index_d += step;
    }
    //   free(space);
}

void PCSR::add_edge(uint32_t src, uint32_t dest, uint32_t value)
{
    // cout << "Adding edge (" << src << "," << dest << ")\n";
    if (value != 0)
    {
        node_t node = nodes[src];
        nodes[src].num_neighbors++;
        nodes[dest].in_degree++;

        edge_t e;
        e.dest = dest;
        e.value = value;

        uint32_t loc_to_add =
            binary_search(&edges, &e, node.beginning + 1, node.end);
        insert(loc_to_add, e, src);
        ++edge_count;
    }
}

void PCSR::add_edge_update(uint32_t src, uint32_t dest, uint32_t value)
{
    if (value != 0)
    {
        node_t node = nodes[src];

        edge_t e;
        e.dest = dest;
        e.value = value;

        uint32_t loc_to_add =
            binary_search(&edges, &e, node.beginning + 1, node.end);
        if (edges.items[loc_to_add].dest == dest)
        {
            edges.items[loc_to_add].value = value;
            return;
        }
        nodes[src].num_neighbors++;
        nodes[dest].in_degree++;
        insert(loc_to_add, e, src);
        ++edge_count;
    }
}

void PCSR::delete_edge(uint32_t src, uint32_t dest)
{
    // cout << "Deleting edge (" << src << "," << dest << ")\n";
    edge_t e;
    e.value = 0;
    e.dest = dest;
    uint32_t loc = binary_search(&edges, &e, nodes[src].beginning + 1, nodes[src].end);

    if (!is_null(edges.items[loc]) && edges.items[loc].dest == dest)
    {
        edges.items[loc].value = 0;
        nodes[src].num_neighbors -= 1;
        nodes[dest].in_degree -= 1;
        --edge_count;
    }
}

uint64_t PCSR::get_n()
{
    return nodes.size();
}

// vector<tuple<uint32_t, uint32_t, uint32_t>> PCSR::get_edges()
// {
//     uint64_t n = get_n();
//     vector<tuple<uint32_t, uint32_t, uint32_t>> output;

//     output.resize(edge_count);
//     int iter = 0;
//     for (int i = 0; i < n; i++)
//     {
//         uint32_t start = nodes[i].beginning;
//         uint32_t end = nodes[i].end;
//         for (int j = start + 1; j < end; j++)
//         {
//             if (!is_null(edges.items[j]))
//             {
//                 output[iter] =
//                     make_tuple(i, edges.items[j].dest, edges.items[j].value);
//                 iter += 1;
//             }
//         }
//     }
//     return output;
// }

void PCSR::print_graph()
{
    int num_vertices = nodes.size();

    // printing the graph matrix column indices
    for (int i = 0; i < num_vertices; ++i)
        printf("   %d", i);

    printf("\n");

    for (int i = 0; i < num_vertices; i++)
    {
        // +1 to avoid sentinel

        // printing the graph matrix row indices
        printf("%d ", i);
        int matrix_index = 0;

        for (uint32_t j = nodes[i].beginning + 1; j < nodes[i].end; j++)
        {
            if (!is_null(edges.items[j]))
            {
                while (matrix_index < edges.items[j].dest)
                {
                    printf("    ");
                    matrix_index++;
                }
                // printf("%03d ", edges.items[j].value);
                printf(" %d  ", edges.items[j].value);
                matrix_index++;
            }
        }
        for (uint32_t j = matrix_index; j < num_vertices; j++)
        {
            printf("    ");
        }
        printf("\n");
    }
}

void PCSR::print_array()
{
    for (int i = 0; i < edges.N; i++)
    {
        if (is_null(edges.items[i]))
        {
            printf("%d-x ", i);
        }
        else if (is_sentinel(edges.items[i]))
        {
            uint32_t value = edges.items[i].value;
            if (value == UINT32_MAX)
            {
                value = 0;
            }
            printf("\n%d-s(%u):(%d, %d) ", i, value, nodes[value].beginning,
                   nodes[value].end);
        }
        else
        {
            printf("%d-(%d, %u) ", i, edges.items[i].dest, edges.items[i].value);
        }
    }
    printf("\n\n");
}

std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t> PCSR::get_csr_ptrs(std::vector<int> eids)
{

    // if (edge_count != eids.size())
    // {
    //     cout << "🍎🍎🍎 The eids(" << eids.size() << ") and edge_count(" << edge_count << ") [col_indices.size() = " << edges.items.size() << "] DONT match\n"
    //          << flush;
    // }

    // we are building a compressed CSR arrays without
    // the -1 values which indicates an empty edge
    thrust::host_vector<int> row_offset;
    thrust::host_vector<int> column_indices;

    int row_offset_size = nodes.size() + 1;
    int column_indices_size = edges.items.size();

    // first element of row offset is always zero
    row_offset.resize(row_offset_size);
    column_indices.resize(edge_count);
    row_offset[0] = 0;

    for (int i = 0; i < row_offset_size - 1; ++i)
    {
        row_offset[i + 1] = nodes[i].num_neighbors + row_offset[i];
    }

    int iter = 0;
    for (int i = 0; i < column_indices_size; ++i)
    {
        if (!is_sentinel(edges.items[i]) && !is_null(edges.items[i]))
        {
            column_indices[iter] = edges.items[i].dest;
            iter += 1;
        }
    }

    row_offset_device = row_offset;
    column_indices_device = column_indices;
    eids_device.resize(edge_count);

    if (eids.size() == 0)
        thrust::sequence(eids_device.begin(), eids_device.end());
    else
        thrust::copy(eids.begin(), eids.end(), eids_device.begin());

    std::tuple<std::uintptr_t, std::uintptr_t, std::uintptr_t> t;
    std::get<0>(t) = (std::uintptr_t)RAW_PTR(row_offset_device);
    std::get<1>(t) = (std::uintptr_t)RAW_PTR(column_indices_device);
    std::get<2>(t) = (std::uintptr_t)RAW_PTR(eids_device);
    return t;
}

// void PCSR::label_edges()
// {

//     uint32_t column_indices_size = edges.items.size();
//     uint32_t counter = 1;
//     for (int i = 0; i < column_indices_size; ++i)
//     {
//         if (!is_sentinel(edges.items[i]) && !is_null(edges.items[i]))
//         {
//             edges.items[i].value = counter;
//             ++counter;
//         }
//     }
// }

// uint32_t PCSR::find_edge_id(uint32_t src, uint32_t dest)
// {
//     edge_t e;
//     e.dest = dest;
//     // setting a random number
//     e.value = 0;

//     uint32_t loc = binary_search(&edges, &e, nodes[src].beginning + 1, nodes[src].end);
//     return edges.items[loc].value;
// }

// std::tuple<uint32_t, uint32_t> PCSR::get_graph_attr()
// {
//     std::tuple<uint32_t, uint32_t> t;
//     std::get<0>(t) = nodes.size();
//     std::get<1>(t) = edge_count;
//     return t;
// }

// void copy_label_edges(PCSR &pcsr, PCSR &ref_pcsr)
// {
//     uint64_t n = pcsr.get_n();

//     for (int i = 0; i < n; i++)
//     {
//         uint32_t start = pcsr.nodes[i].beginning;
//         uint32_t end = pcsr.nodes[i].end;
//         for (int j = start + 1; j < end; j++)
//         {
//             if (!is_null(pcsr.edges.items[j]))
//             {
//                 // Searching for the edge_id of reverse edge from reference_pcsr
//                 pcsr.edges.items[j].value = ref_pcsr.find_edge_id(pcsr.edges.items[j].dest, i);
//             }
//         }
//     }
// }

void PCSR::edge_update_list(std::vector<std::tuple<uint32_t, uint32_t>> edge_list, bool is_delete = false, bool is_reverse_edge = false)
{
    // cout << "📦📦📦 Edge Update list" << endl
    //      << flush;
    for (auto &edge : edge_list)
    {
        uint32_t src = (is_reverse_edge == true) ? std::get<1>(edge) : std::get<0>(edge);
        uint32_t dst = (is_reverse_edge == true) ? std::get<0>(edge) : std::get<1>(edge);

        if (is_delete)
            delete_edge(src, dst);
        else
            add_edge(src, dst, 1);
    }
}

void build_reverse_pcsr(PCSR &pcsr, PCSR &ref_pcsr)
{
    // cout << "💄💄💄 Building Reverse PCSR" << endl
    //      << flush;
    uint64_t n = ref_pcsr.get_n();
    for (int i = 0; i < n; i++)
    {
        uint32_t start = ref_pcsr.nodes[i].beginning;
        uint32_t end = ref_pcsr.nodes[i].end;
        for (int j = start + 1; j < end; j++)
        {
            if (!is_sentinel(ref_pcsr.edges.items[j]) && !is_null(ref_pcsr.edges.items[j]))
            {
                pcsr.add_edge(ref_pcsr.edges.items[j].dest, i, ref_pcsr.edges.items[j].value);
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////

// PCSR Python Module

PYBIND11_MODULE(pcsr, m)
{
    m.doc() = "PCSR Dynamic Graph Representation";
    // m.def("copy_label_edges", &copy_label_edges, "Label edges of a PCSR based on another PCSR");
    m.def("build_reverse_pcsr", &build_reverse_pcsr, "Builds the reverse PCSR based on another PCSR");

    py::class_<node_t>(m, "Node")
        .def(py::init<int, int, int>(), py::arg("beg") = 0, py::arg("_end") = 0, py::arg("num_neigh") = 0)
        .def_readwrite("beginning", &_node::beginning)
        .def_readwrite("end", &_node::end)
        .def_readwrite("num_neighbors", &_node::num_neighbors)
        .def_readwrite("in_degree", &_node::in_degree);

    py::class_<edge_t>(m, "Edge")
        .def(py::init<int, int>(), py::arg("_dest") = 0, py::arg("_value") = 0)
        .def_readwrite("dest", &edge_t::dest)
        .def_readwrite("value", &edge_t::value);

    py::class_<edge_list_t>(m, "EdgeList")
        .def(py::init<>())
        .def_readwrite("N", &edge_list_t::N)
        .def_readwrite("H", &edge_list_t::H)
        .def_readwrite("logN", &edge_list_t::logN)
        .def_readwrite("items", &edge_list_t::items);

    py::class_<pair_int>(m, "PairInt")
        .def(py::init<int, int>())
        .def_readwrite("x", &pair_int::x)
        .def_readwrite("y", &pair_int::y);

    py::class_<pair_double>(m, "PairDouble")
        .def(py::init<double, double>())
        .def_readwrite("x", &pair_double::x)
        .def_readwrite("y", &pair_double::y);

    py::class_<PCSR>(m, "PCSR")
        .def(py::init<int>(), py::arg("init_n") = 0)
        .def("init_graph", &PCSR::init_graph)
        .def("add_node", &PCSR::add_node)
        .def("insert", &PCSR::insert)
        .def("double_list", &PCSR::double_list)
        .def("slide_right", &PCSR::slide_right)
        .def("slide_left", &PCSR::slide_left)
        .def("fix_sentinel", &PCSR::fix_sentinel)
        .def("redistribute", &PCSR::redistribute)
        .def("print_graph", &PCSR::print_graph)
        .def("add_edge", &PCSR::add_edge)
        .def("add_edge_update", &PCSR::add_edge_update)
        .def("edge_update_list", &PCSR::edge_update_list, py::arg("edge_list"), py::arg("is_delete") = false, py::arg("is_reverse_edge") = false)
        // .def("label_edges", &PCSR::label_edges, "Creates edge labels for the current GPMA")
        .def("delete_edge", &PCSR::delete_edge)
        .def("get_n", &PCSR::get_n)
        // .def("get_edges", &PCSR::get_edges)
        .def("print_array", &PCSR::print_array)
        .def("get_csr_ptrs", &PCSR::get_csr_ptrs, py::arg("eids"))
        // .def("get_graph_attr", &PCSR::get_graph_attr)
        // .def("find_edge_id", &PCSR::find_edge_id)
        .def_readwrite("nodes", &PCSR::nodes)
        .def_readwrite("edges", &PCSR::edges)
        .def_readwrite("edge_count", &PCSR::edge_count)
        .def("__copy__", [](const PCSR &self)
             { return PCSR(self); })
        .def(
            "__deepcopy__", [](const PCSR &self, py::dict)
            { return PCSR(self); },
            "memo"_a);
}

// /usr/local/cuda-11.7/bin/nvcc $(python3 -m pybind11 --includes) -shared -rdc=true --compiler-options '-fPIC'  -D__CDPRT_SUPPRESS_SYNC_DEPRECATION_WARNING -o pcsr.so pcsr.cu