#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <hipcub/hipcub.hpp>

int main()
{
    thrust::host_vector<int> vec_1(500, 10);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    thrust::device_vector<int> vec_2 = vec_1;
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("\n⏱️ Time taken: %.3f\n", milliseconds);

    thrust::host_vector<int> vec_3(500, 10);
    thrust::device_vector<int> d_vec_3(500);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < 500; ++i)
    {
        d_vec_3[i] = vec_3[i];
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("\n⏱️ Time taken: %.3f\n", milliseconds);
}