
#include <hip/hip_runtime.h>
extern "C" __global__ void K4(
  float *Vhinb, float *Vnormcen, float *Vnorminb, float *Vweight, float *V23, 
  unsigned int *row_offsets,
  unsigned int *eids,
  unsigned long long *column_indices,
  int num_nodes,
  int max_dimx,
  int max_dimy,
  int thrs_per_group,
  int nodes_per_block) {

    int dst_id = nodes_per_block*blockIdx.x + threadIdx.x/thrs_per_group;

    if (dst_id < num_nodes) {

        int feat_len = max_dimx * max_dimy;
        unsigned int beg = __ldg(row_offsets + dst_id);
        unsigned int end = __ldg(row_offsets + dst_id + 1);
        int tx = threadIdx.x % thrs_per_group;

        for (; tx<feat_len; tx+=blockDim.x) {

            float V22_tmp = 0;
            int offset3 = dst_id * 1 + tx/32;int offset4 = dst_id * 32 + tx;

            for (int e=beg;e<end;++e) {

                unsigned long long src_id = __ldg(column_indices + e);

                // GPMA indexes edges starting from 1
                // Seastar requires edgs to be indexed from 0
                double eid = __ldg(eids + e) - 1;
                
                unsigned long long mask = (unsigned long long)dst_id << 32;
                unsigned int dst_check = (src_id - mask);
                src_id = (src_id - mask);

                if(dst_check != 0xFFFFFFFF && eid != 0){
                    int offset0 = src_id * 1 + tx/32;int offset1 = src_id * 32 + tx;int offset2 = eid * 1 + tx/32;

                    
                    
                    float V20_tmp = Vnorminb[offset0]*Vhinb[offset1];
                    
                    
                    
                    float V21_tmp = V20_tmp*Vweight[offset2];
                    
                    

                    
                    V22_tmp += V21_tmp;
                    
                        
                }
            }

            
            
            

            
            
            float V23_tmp = V22_tmp*Vnormcen[offset3];
            V23[offset4] = V23_tmp;
            
        }
    }
}extern "C" __global__ void K5(
  float *V24, float *Vnormcen, float *Vnorminb, float *Vweight, float *V29, 
  unsigned int *row_offsets,
  unsigned int *eids,
  unsigned long long *column_indices,
  int num_nodes,
  int max_dimx,
  int max_dimy,
  int thrs_per_group,
  int nodes_per_block) {

    int src_id = nodes_per_block*blockIdx.x + threadIdx.x/thrs_per_group;

    if (src_id < num_nodes) {

        int feat_len = max_dimx * max_dimy;
        unsigned int beg = __ldg(row_offsets + src_id);
        unsigned int end = __ldg(row_offsets + src_id + 1);
        int tx = threadIdx.x % thrs_per_group;

        for (; tx<feat_len; tx+=blockDim.x) {

            float V28_tmp = 0;
            int offset3 = src_id * 1 + tx/32;int offset4 = src_id * 32 + tx;

            for (int e=beg;e<end;++e) {

                unsigned long long dst_id = __ldg(column_indices + e);

                // GPMA indexes edges starting from 1
                // Seastar requires edgs to be indexed from 0
                double eid = __ldg(eids + e) - 1;
                
                unsigned long long mask = (unsigned long long)src_id << 32;
                unsigned int dst_check = (dst_id - mask);
                dst_id = (dst_id - mask);

                if(dst_check != 0xFFFFFFFF && eid != 0){
                    int offset0 = dst_id * 1 + tx/32;int offset1 = dst_id * 32 + tx;int offset2 = eid * 1 + tx/32;

                    
                    
                    float V25_tmp = V24[offset1]*Vnormcen[offset0];
                    
                    
                    
                    float V27_tmp = V25_tmp*Vweight[offset2];
                    
                    

                    
                    V28_tmp += V27_tmp;
                    
                        
                }
            }

            
            
            

            
            
            float V29_tmp = V28_tmp*Vnorminb[offset3];
            V29[offset4] = V29_tmp;
            
        }
    }
}